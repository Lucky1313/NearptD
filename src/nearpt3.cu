#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/copy.h>

#include <algorithm>
#include <boost/multi_array.hpp> 
#include <iomanip>
#include <iostream>
#include <fstream>
#include <math.h>
#include <sys/resource.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <sys/times.h>
#include <sys/types.h>
#include <time.h>
#include <unistd.h>

#include "grid.cu"

using namespace std;
using boost::array;

namespace nearpt3 {
  
	double ng_factor = 1.6;

  // cellsearchorder:
  // First 3 elements of each row:  the order in which to search (one 48th-ant of the) cells adjacent to the current cell.
  // 4th element:   where, in cellsearchorder, to stop searching after the first point is found.
  const int cellsearchorder[] = {
#include "cellsearchorder"
  };
  // Number of cells in cellsearchorder (before expanding symmetries).
  const int ncellsearchorder = 
    sizeof(nearpt3::cellsearchorder) / sizeof(nearpt3::cellsearchorder[0])/4;

  template<typename Coord_T> Grid_T<Coord_T>*
  Preprocess(const int nfixpts, Points_Vector<Coord_T>* pts) {
    typedef typename Grid_T<Coord_T>::Coord_Tuple Coord_Tuple;
    typedef typename Grid_T<Coord_T>::Coord_Iterator_Tuple Coord_Iterator_Tuple;
    
    Grid_T<Coord_T> *g;
    g = new Grid_T<Coord_T>;
    g->nfixpts = nfixpts;
    int &ng = g->ng;
    ng = static_cast<int> (ng_factor * cbrt(static_cast<double>(nfixpts)));

    ng = min(2000, max(1, ng));
    g->ng3 = ng * ng * ng;
    g->pts = pts;

    for (int i=1; i<ncellsearchorder; ++i)
      if (nearpt3::cellsearchorder[(i-1)*4+3] > nearpt3::cellsearchorder[i*4+3]) 
        throw "cellsearchorder is not monotonic";

    thrust::pair<array<Coord_T,3>, array<Coord_T,3> > minmax = pts->minmax();
    array<Coord_T,3> lo = thrust::get<0>(minmax);
    array<Coord_T,3> hi = thrust::get<1>(minmax);

    #ifdef DEBUG
    cout << "Min/Max" << endl;
    cout << lo[0] << ", " << lo[1] << ", " << lo[2] << endl;
    cout << hi[0] << ", " << hi[1] << ", " << hi[2] << endl;
    #endif

    array<double,3> s;
    for (int i=0; i<3; ++i) {
      s[i] = 0.99 * ng / static_cast<double>(hi[i] - lo[i]);
    }
    g->r_cell = min(min(s[0], s[1]), s[2]);

    for(int i=0; i<3; i++) {
      g->d_cell[i] = ((ng-1)-(lo[i]+hi[i])*g->r_cell) * 0.5;
      g->d[i] = g->d_cell[i];
    }

    // Create device vectors (Must be before functors)
    g->base = thrust::device_vector<int>(g->ng3+1, 1);
    g->cells = thrust::device_vector<int>(g->nfixpts);
    g->cell_indices = thrust::device_vector<int>(g->nfixpts);
    thrust::sequence(g->cell_indices.begin(), g->cell_indices.end());

    g->cellsearch = thrust::device_vector<int>(cellsearchorder, cellsearchorder+ncellsearchorder*4);

    // Define functors
    g->check_cell = check_cell_functor(g->ng);
    g->clip_cell = clip_cell_functor(g->ng);
    g->cell_containing_point = cell_containing_point_functor<Coord_Tuple>(g->r_cell, g->d[0], g->d[1], g->d[2]);
    g->cell_to_id = cell_to_id_functor(g->ng);
    g->point_to_id = point_to_id_functor<Coord_Tuple>(g->cell_containing_point, g->cell_to_id);
    g->num_points_in_cell_id = num_points_in_cell_id_functor(g->base.data());
    g->query_cell = query_cell_functor<Coord_T>(g->num_points_in_cell_id,
                                                g->point_to_id,
                                                g->base.data(),
                                                g->cells.data(),
                                                pts->get_ptrs());
    g->fast_query = fast_query_functor<Coord_T>(g->clip_cell,
                                                g->cell_containing_point,
                                                g->query_cell);
    g->slow_query = slow_query_functor<Coord_T>(ncellsearchorder,
                                                g->cellsearch.data(),
                                                g->check_cell,
                                                g->cell_containing_point,
                                                g->query_cell);
    
    
    #ifdef DEBUG
    cout << "Grid info:";
    cout << "\nng: " << g->ng;
    cout << "\nng3: " << g->ng3;
    cout << "\ns: (" << s[0] << ", " << s[1] << ", " << s[2] << ")";
    cout << "\nr_cell: " << g->r_cell;
    cout << "\nd_cell: (" << g->d_cell[0] << ", " << g->d_cell[1] << ", " << g->d_cell[2] << ")";
    cout << endl;
    #endif
    
    // Calculate cell id from point
    thrust::transform(pts->begin(), pts->end(), g->cells.begin(), g->point_to_id);
    
    // Ensure no cells are -1 (outside range)
    if (thrust::find(g->cells.begin(), g->cells.end(), -1) != g->cells.end()) {
      throw "Bad cell";
    }

    // Keep track of the indices of cells during sorting
    //thrust::stable_sort_by_key(g->cells.begin(), g->cells.end(), g->cell_indices.begin());
    thrust::sort_by_key(g->cells.begin(), g->cells.end(), g->cell_indices.begin());
    
    // Taken from thrust histogram example
    thrust::counting_iterator<int> count(0);
    thrust::lower_bound(g->cells.begin(), g->cells.end(),
                        count, count + g->ng3 + 1,
                        g->base.begin());

    #ifdef DEBUG
    cout << "Base: [";
    thrust::copy(g->base.begin(), g->base.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    if (g->base[g->ng3] != nfixpts) {
      cout << "ERROR: Internal inconsistency; wrong " << PRINTN(g->base[g->ng3]);
      throw "Internal inconsistency";
    }

    // Transform iterator to compute point ids then permutation iterator to get value from base
    typedef thrust::transform_iterator<point_to_id_functor<Coord_Tuple>, Coord_Iterator_Tuple> ptid_itr;
    ptid_itr ptid_begin(pts->begin(), g->point_to_id);
    ptid_itr ptid_end(pts->end(), g->point_to_id);

    typedef thrust::device_vector<int>::iterator IntItr;
    typedef thrust::permutation_iterator<IntItr, ptid_itr> PermItr;
    PermItr cbbegin(g->base.begin(), ptid_begin);

    // Exclusive scan by key to get count for number of points in cell
    thrust::constant_iterator<int> one(1);
    thrust::exclusive_scan_by_key(g->cells.begin(), g->cells.end(), one, g->cells.begin());

    // 'Undo' previous sort to have an increasing point count per cell
    //thrust::stable_sort_by_key(g->cell_indices.begin(), g->cell_indices.end(), g->cells.begin());
    thrust::sort_by_key(g->cell_indices.begin(), g->cell_indices.end(), g->cells.begin());

    // Offset calculated base indices from permutation iterator by point per cell count
    thrust::plus<int> plus_op;
    thrust::transform(g->cells.begin(), g->cells.end(), cbbegin, g->cell_indices.begin(), plus_op);

    // Fill cells with increasing count
    thrust::sequence(g->cells.begin(), g->cells.end());

    // Reorder indices by offset base indices
    //thrust::stable_sort_by_key(g->cell_indices.begin(), g->cell_indices.end(), g->cells.begin());
    thrust::sort_by_key(g->cell_indices.begin(), g->cell_indices.end(), g->cells.begin());
    
    #ifdef DEBUG
    cout << "Cells: [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    #ifdef STATS
    g->Num_Points_Per_Cell.resize(g->ng3, 0);
    g->Num_Cells_Searched.resize(g->Max_Cells_Searched+1, 0);
    g->Num_Points_Checked.resize(g->Max_Points_Checked+1, 0);

    g->Min_Points_Per_Cell = -1;
    g->Max_Points_Per_Cell = -1;
    g->Avg_Points_Per_Cell = -1;
    g->Num_Fast_Queries = 0;
    g->Num_Slow_Queries = 0;
    g->Num_Exhaustive_Queries = 0;
    g->Total_Cells_Searched = 0;
    g->Total_Points_Checked = 0;
    g->Points_Checked = 0;

    thrust::adjacent_difference(g->base.begin(), g->base.end(), g->Num_Points_Per_Cell.begin());
    g->Min_Points_Per_Cell = *thrust::min_element(g->Num_Points_Per_Cell.begin()+1,
                                                  g->Num_Points_Per_Cell.end());
    g->Max_Points_Per_Cell = *thrust::max_element(g->Num_Points_Per_Cell.begin()+1,
                                                  g->Num_Points_Per_Cell.end());
    g->Avg_Points_Per_Cell = static_cast<float>(nfixpts) / static_cast<float>(g->ng3);
    #endif

    return g;
  }

  template<typename Coord_T>
  void Query(Grid_T<Coord_T>* g, Points_Vector<Coord_T>* q, thrust::device_vector<int> *closest) {
    typedef typename Grid_T<Coord_T>::Coord_Tuple Coord_Tuple;
    typedef typename Grid_T<Coord_T>::Coord_Iterator_Tuple Coord_Iterator_Tuple;
    
    const int nqpts(q->size());
    thrust::device_vector<int> qindices(nqpts);
    thrust::sequence(qindices.begin(), qindices.end());

    // Calculate id for query points
    thrust::device_vector<int> qcells(nqpts);
    thrust::transform(q->begin(), q->end(), qcells.begin(), g->point_to_id);

    #ifdef DEBUG
    cout << "Query IDs: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    thrust::transform(qcells.begin(), qcells.end(), qcells.begin(), g->num_points_in_cell_id);
    
    #ifdef DEBUG
    cout << "Number of points in cells: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    // Partition by number of points in a cell, so all non-empty ones are together and can be iterated over
    typedef thrust::device_vector<int>::iterator IntItr;
    typedef thrust::zip_iterator<thrust::tuple<IntItr, IntItr> > ZipItr;
    greater_functor<thrust::tuple<int, int> > greater_zero(0);
    //less_functor<int> less_zero(0);
    // Is greater or less faster?
    // Zip iterator to reorder the cells and indices
    ZipItr zbegin(thrust::make_zip_iterator(thrust::make_tuple(qcells.begin(), qindices.begin())));
    ZipItr zend(thrust::make_zip_iterator(thrust::make_tuple(qcells.end(), qindices.end())));
    ZipItr itr = thrust::partition(zbegin, zend, greater_zero);
    //itr = thrust::partition(query_indices.begin(), query_indices.end(), less_zero);
    
    #ifdef DEBUG
    cout << "Partition: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    cout << "Indices: [";
    thrust::copy(qindices.begin(), qindices.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    // If greater than zero, range for fast is begin to itr, if less, itr to end
    // Do fast case query on all query points that have points in the cell
    // Permutation from indices to actual points
    typedef thrust::permutation_iterator<Coord_Iterator_Tuple, IntItr> PermItr;
    PermItr qbegin(q->begin(), qindices.begin());
    thrust::transform(qbegin, qbegin + (itr - zbegin), qcells.begin(), g->fast_query);

    #ifdef DEBUG
    cout << "Fast from (" << 0 << ", " << (itr - zbegin) << ")" << endl;
    cout << "Fast Query Results: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    // Slow case query
    thrust::transform(qbegin + (itr - zbegin), qbegin + nqpts,
                      qcells.begin() + (itr - zbegin), g->slow_query);
    //cout << slowtest(g, *(qbegin+(itr-zbegin))) << endl;
    
    #ifdef DEBUG
    cout << "Slow from (" << (itr - zbegin) << ", " << nqpts << ")" << endl;
    cout << "Slow Query Results: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    // Any slow case queries that returned -1 need to be done exhaustively.
    // It will be faster to do parallel search over all points, rather than parallel exhaustive searches
    greater_functor<thrust::tuple<int, int> > positive(-1);
    itr = thrust::partition(zbegin, zend, positive);

    #ifdef DEBUG
    cout << "Repartition: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    for (int i=(itr - zbegin); i<nqpts; ++i) {
      qcells[i] = g->exhaustive_query((*q)[i]);
    }
    
    #ifdef DEBUG
    cout << "Exhaustive from (" << (itr - zbegin) << ", " << nqpts << ")" << endl;
    cout << "Exhaustive Query Results: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    // Resort query points by index, could be skipped if order of queries doesn't matter
    thrust::sort_by_key(qindices.begin(), qindices.end(), qcells.begin());
    
    #ifdef DEBUG
    cout << "Resorted Query results: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    // Copy results to the closest points array
    thrust::copy(qcells.begin(), qcells.end(), closest->begin());
  }
};
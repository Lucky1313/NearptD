#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/copy.h>

#include <algorithm>
#include <boost/multi_array.hpp> 
#include <iomanip>
#include <iostream>
#include <fstream>
#include <math.h>
#include <sys/resource.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <sys/times.h>
#include <sys/types.h>
#include <time.h>
#include <unistd.h>

#include "grid.cu"

using namespace std;
using boost::array;

#define DEBUG

namespace nearpt3 {
	double ng_factor = 1.6;

  // cellsearchorder:
  // First 3 elements of each row:  the order in which to search (one 48th-ant of the) cells adjacent to the current cell.
  // 4th element:   where, in cellsearchorder, to stop searching after the first point is found.
  const static int  cellsearchorder[][4] = {
#include "cellsearchorder"
  };
  // Number of cells in cellsearchorder (before expanding symmetries).
  const static int ncellsearchorder = 
    sizeof(nearpt3::cellsearchorder) / sizeof(nearpt3::cellsearchorder[0][0])/4;

  

  void write(ostream &o, const Cell3& c) {
    o << '(' << c[0] << ',' << c[1] << ',' << c[2] << ") ";
  }
  

  template<typename Coord_T> Grid_T<Coord_T>*
  Preprocess(const int nfixpts, Points_T<Coord_T>* pts) {
    typedef thrust::tuple<Coord_T, Coord_T, Coord_T> Coord3;
    
    Grid_T<Coord_T> *g;
    g = new Grid_T<Coord_T>;
    g->nfixpts = nfixpts;
    int &ng = g->ng;
    ng = static_cast<int> (ng_factor * cbrt(static_cast<double>(nfixpts)));

    ng = min(2000, max(1, ng));
    g->ng3 = ng * ng * ng;
    g->pts = pts;

    for (int i=1; i<ncellsearchorder; ++i)
      if (nearpt3::cellsearchorder[i-1][3] > nearpt3::cellsearchorder[i][3]) 
        throw "cellsearchorder is not monotonic";

    thrust::pair<array<Coord_T,3>, array<Coord_T,3> > minmax = pts->minmax();
    array<Coord_T,3> lo = thrust::get<0>(minmax);
    array<Coord_T,3> hi = thrust::get<1>(minmax);

    #ifdef DEBUG
    cout << "Min/Max" << endl;
    cout << lo[0] << ", " << lo[1] << ", " << lo[2] << endl;
    cout << hi[0] << ", " << hi[1] << ", " << hi[2] << endl;
    #endif

    array<double,3> s;
    for (int i=0; i<3; ++i) {
      s[i] = 0.99 * ng / static_cast<double>(hi[i] - lo[i]);
    }
    g->r_cell = min(min(s[0], s[1]), s[2]);

    for(int i=0; i<3; i++) {
      g->d_cell[i] = ((ng-1)-(lo[i]+hi[i])*g->r_cell) * 0.5;
    }

    #ifdef DEBUG
    cout << "Grid info:";
    cout << "\nng: " << g->ng;
    cout << "\nng3: " << g->ng3;
    cout << "\ns: (" << s[0] << ", " << s[1] << ", " << s[2] << ")";
    cout << "\nr_cell: " << g->r_cell;
    cout << "\nd_cell: " << g->d_cell[0] << ", " << g->d_cell[1] << ", " << g->d_cell[2] << ")";
    cout << endl;
    #endif

    g->base = thrust::device_vector<int>(g->ng3+1, 1);
    g->cells = thrust::device_vector<int>(g->nfixpts);

    // Calculate cell id from point
    thrust::transform(pts->begin(), pts->end(), g->cells.begin(),
                      point_to_id_functor<Coord3>(g->ng, g->r_cell, g->d_cell[0],
                                                  g->d_cell[1], g->d_cell[2]));

    #ifdef DEBUG
    cout << "Cell IDs (cells): [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    // Ensure no cells are -1 (outside range)
    if (thrust::find(g->cells.begin(), g->cells.end(), -1) != g->cells.end()) {
      throw "Bad cell";
    }

    thrust::sort(g->cells.begin(), g->cells.end());

    #ifdef DEBUG
    cout << "Sorted Cell IDs (cells): [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    // Taken from thrust histogram example
    thrust::counting_iterator<int> count(0);
    thrust::lower_bound(g->cells.begin(), g->cells.end(),
                        count, count + g->ng3 + 1,
                        g->base.begin());

    #ifdef DEBUG
    cout << "Count: " << *count << endl;
    cout << "Lower bound (base): [";
    thrust::copy(g->base.begin(), g->base.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    if (g->base[g->ng3] != nfixpts) {
      cout << "ERROR: Internal inconsistency; wrong " << PRINTN(g->base[g->ng3]);
      throw "Internal inconsistency";
    }
    
    thrust::fill(g->cells.begin(), g->cells.end(), 0);

    // SERIAL
    for (int n=0; n<g->nfixpts; ++n) {
      const int ic(g->point_to_id(n));
      const int pitc = g->cells[g->base[ic+1]-1]++;
      g->cells[g->base[ic]+pitc] = n;
    }
    /*
    thrust::transform(pts->begin(), pts->end(), g->cells.begin(),
                      point_to_id_functor<Coord3>(g->ng, g->r_cell, g->d_cell[0],
                                                  g->d_cell[1], g->d_cell[2]));
    thrust::stable_sort_by_key();
    */
    #ifdef DEBUG
    cout << "Iterative (cells): [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    return g;
  }

  template<typename Coord_T> int
  Query(Grid_T<Coord_T>* g, const array<Coord_T, 3> q) {

    int closestpt(g->Query_Fast_Case(q));
    if (closestpt>=0) {
      return closestpt;
    }

    Cell3 querycell(g->Compute_Cell_Containing_Point(q));

    double dist(numeric_limits<double>::max());
    int closecell(-1);
    int goodsortnum;
    bool foundit(false);
    int nstop(ncellsearchorder);
    
    for (int isort=0; isort<nstop; ++isort) {
      int thisclosest;
      double thisdist;
      Cell3 s (cellsearchorder[isort][0], cellsearchorder[isort][1], 
	       cellsearchorder[isort][2]);

      for (int isign=0; isign<8; isign++) {      // Iterate over all combinations of signs;
        static const int sign3[8][3] = {{1,1,1},{1,1,-1},{1,-1,1},{1,-1,-1},
                                        {-1,1,1},{-1,1,-1},{-1,-1,1},{-1,-1,-1}};
        if (s[0]==0 && sign3[isign][0]== -1) continue;
        if (s[1]==0 && sign3[isign][1]== -1) continue;
        if (s[2]==0 && sign3[isign][2]== -1) continue;

        const Cell3 s2(s*sign3[isign]);

        for (int iperm=0; iperm<6; iperm++) {   // Iterate over all permutations of coordinates.
          switch (iperm) {
          case 1:
            if (s[1]==s[2]) continue;
            break;
          case 2: 
            if (s[0]==s[1]) continue;
            break;
          case 3:
          case 4:
            if (s[0]==s[1] && s[0]==s[2]) continue;
            break;
          case 5:
            if (s[0]==s[2]) continue;
            break;
          }
          static const int perm3[6][3] = {{0,1,2},{0,2,1},{1,0,2},{1,2,0},{2,0,1},{2,1,0}};
          const Cell3 s3(s2[perm3[iperm][0]], s2[perm3[iperm][1]], s2[perm3[iperm][2]]);
          const Cell3 c2(querycell+s3);
          if (!g->check(c2)) continue;  // outside the universe?
          goodsortnum = isort;
          g->querythiscell(c2, q, thisclosest, thisdist);
          if (thisclosest < 0) continue;

          // If two fixed points are the same distance from the query, then return the one with the
          // smallest index.  This removes ambiguities, but complicates the code in several places.
          
          if (thisdist<dist || (thisdist==dist && thisclosest<closestpt)) {
            dist = thisdist;
            closestpt = thisclosest;
            closecell =  g->cellid_to_int(c2);
            if (!foundit) {
              foundit = true;
              nstop = cellsearchorder[isort][3];
              if (nstop >= ncellsearchorder) {
                // It took so long to find any cell with a point that cellsearchorder doesn't have
                // enough cells to be sure of finding the closest point.  Fall back to naive
                // exhaustive searching.
                goto L_end_isort;
              }
            }
          }
        }
      }
    }

  L_end_isort: if (closestpt>=0) {
      return closestpt;
    }
    
    // No nearby points, so exhaustively search over all the fixed points.
    typedef thrust::tuple<Coord_T, Coord_T, Coord_T> Coord3;
    typedef thrust::device_vector<Coord_T> Coord_Vector;
    typedef typename Coord_Vector::iterator Coord_Iterator;
    typedef thrust::tuple<Coord_Iterator, Coord_Iterator, Coord_Iterator> Coord_Iterator_Tuple;
    typedef thrust::zip_iterator<Coord_Iterator_Tuple> Coord_3_Iterator;
    typedef thrust::device_vector<int>::iterator IntItr;
    typedef thrust::transform_iterator<distance2_functor<Coord3>, Coord_3_Iterator> dist2_itr;
    dist2_itr begin(g->pts->begin(),
                    distance2_functor<Coord3>(q[0], q[1], q[2]));
    dist2_itr end(g->pts->end(),
                    distance2_functor<Coord3>(q[0], q[1], q[2]));
    dist2_itr result = thrust::min_element(begin, end);
    closestpt = g->cells[result - begin];
    
    return closestpt;
  }
  
};
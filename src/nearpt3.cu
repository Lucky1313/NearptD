#include "hip/hip_runtime.h"
#pragma once

#include <iostream>

#include <thrust/pair.h>
#include <thrust/tuple.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#ifdef STATS
#include <thrust/adjacent_difference.h>
#endif

#include "point_vector.cu"
#include "grid.cu"
#include "functors.cu"
#include "utils.cpp"

using namespace std;

namespace nearpt3 {
  
	double ng_factor = 1.6;

  // cellsearchorder:
  // First 3 elements of each row:  the order in which to search (one 48th-ant of the) cells adjacent to the current cell.
  // 4th element:   where, in cellsearchorder, to stop searching after the first point is found.
  const int cellsearchorder[] = {
#include "cellsearchorder"
  };
  // Number of cells in cellsearchorder (before expanding symmetries).
  const int ncellsearchorder = 
    sizeof(nearpt3::cellsearchorder) / sizeof(nearpt3::cellsearchorder[0])/4;

  // Process all fixed points into a uniform grid
  template<typename Coord_T> Grid_T<Coord_T>*
  Preprocess(const int nfixpts, Point_Vector<Coord_T>* pts) {
    // Typedefs derived from Grid class
    typedef typename Grid_T<Coord_T>::Coord_Tuple Coord_Tuple;
    typedef typename Grid_T<Coord_T>::Coord_Iterator_Tuple Coord_Iterator_Tuple;
    
    Grid_T<Coord_T> *g;
    g = new Grid_T<Coord_T>;
    
    g->nfixpts = nfixpts;
    int &ng = g->ng;
    ng = static_cast<int> (ng_factor * cbrt(static_cast<double>(nfixpts)));

    ng = min(2000, max(1, ng));
    g->ng3 = ng * ng * ng;
    g->pts = pts;

    // Ensure monotonic cell search order
    for (int i=1; i<ncellsearchorder; ++i)
      if (nearpt3::cellsearchorder[(i-1)*4+3] > nearpt3::cellsearchorder[i*4+3]) 
        throw "cellsearchorder is not monotonic";

    // Get min and max of data range
    thrust::pair<Coord_Tuple, Coord_Tuple> minmax = pts->minmax();
    Coord_Tuple lo = thrust::get<0>(minmax);
    Coord_Tuple hi = thrust::get<1>(minmax);

    #ifdef DEBUG
    cout << "Min/Max" << endl;
    cout << thrust::get<0>(lo) << ", " << thrust::get<1>(lo) << ", " << thrust::get<2>(lo) << endl;
    cout << thrust::get<0>(hi) << ", " << thrust::get<1>(hi) << ", " << thrust::get<2>(hi) << endl;
    #endif

    Double_Tuple s(thrust::make_tuple(0.99 * ng / static_cast<double>(thrust::get<0>(hi) - thrust::get<0>(lo)),
                                      0.99 * ng / static_cast<double>(thrust::get<1>(hi) - thrust::get<1>(lo)),
                                      0.99 * ng / static_cast<double>(thrust::get<2>(hi) - thrust::get<2>(lo))));
    
    g->r_cell = min(min(thrust::get<0>(s), thrust::get<1>(s)), thrust::get<2>(s));
    g->d_cell = thrust::make_tuple(((ng-1)-(thrust::get<0>(lo)+thrust::get<0>(hi))*g->r_cell) * 0.5,
                                   ((ng-1)-(thrust::get<1>(lo)+thrust::get<1>(hi))*g->r_cell) * 0.5,
                                   ((ng-1)-(thrust::get<2>(lo)+thrust::get<2>(hi))*g->r_cell) * 0.5);

    // Create device vectors (Must be before functors)
    g->base = thrust::device_vector<int>(g->ng3+1, 1);
    g->cells = thrust::device_vector<int>(g->nfixpts);

    // Copy cell search order to device memory
    g->cellsearch = thrust::device_vector<int>(cellsearchorder, cellsearchorder+ncellsearchorder*4);

    // Define grid functors
    g->check_cell = check_cell_functor(g->ng);
    g->clip_cell = clip_cell_functor(g->ng);
    g->cell_containing_point = cell_containing_point_functor<Coord_Tuple>(g->r_cell, g->d_cell);
    g->cell_to_id = cell_to_id_functor(g->ng);
    g->point_to_id = point_to_id_functor<Coord_Tuple>(g->cell_containing_point, g->cell_to_id);
    g->num_points_in_cell_id = num_points_in_cell_id_functor(g->base.data());
    g->query_cell = query_cell_functor<Coord_T>(g->num_points_in_cell_id,
                                                g->point_to_id,
                                                g->base.data(),
                                                g->cells.data(),
                                                pts->get_ptrs());
    g->fast_query = fast_query_functor<Coord_T>(g->clip_cell,
                                                g->cell_containing_point,
                                                g->query_cell);
    g->slow_query = slow_query_functor<Coord_T>(ncellsearchorder,
                                                g->cellsearch.data(),
                                                g->check_cell,
                                                g->cell_containing_point,
                                                g->query_cell);
    
    #ifdef DEBUG
    cout << "Grid info:";
    cout << "\nng: " << g->ng;
    cout << "\nng3: " << g->ng3;
    cout << "\ns: (" << thrust::get<0>(s) << ", " << thrust::get<1>(s) << ", " << thrust::get<2>(s) << ")";
    cout << "\nr_cell: " << g->r_cell;
    cout << "\nd_cell: (" << thrust::get<0>(g->d_cell) << ", " << thrust::get<1>(g->d_cell) << ", " << thrust::get<2>(g->d_cell) << ")";
    cout << endl;
    #endif

    // Index of cells to safely reorder cells
    thrust::device_vector<int> cell_indices(g->nfixpts);
    thrust::sequence(cell_indices.begin(), cell_indices.end());
    
    // Calculate cell id from point
    thrust::transform(pts->begin(), pts->end(), g->cells.begin(), g->point_to_id);
    
    // Ensure no cells are -1 (outside range)
    if (thrust::find(g->cells.begin(), g->cells.end(), -1) != g->cells.end()) {
      throw "Bad cell";
    }

    // Keep track of the indices of cells during sorting
    thrust::stable_sort_by_key(g->cells.begin(), g->cells.end(), cell_indices.begin());
    //thrust::sort_by_key(g->cells.begin(), g->cells.end(), cell_indices.begin());
    
    // Taken from thrust histogram example
    thrust::counting_iterator<int> count(0);
    thrust::lower_bound(g->cells.begin(), g->cells.end(),
                        count, count + g->ng3 + 1,
                        g->base.begin());

    #ifdef DEBUG
    cout << "Base: [";
    thrust::copy(g->base.begin(), g->base.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    if (g->base[g->ng3] != nfixpts) {
      cout << "ERROR: Internal inconsistency; wrong point count: " << g->base[g->ng3];
      throw "Internal inconsistency";
    }

    // Transform iterator to compute point ids 
    typedef thrust::transform_iterator<point_to_id_functor<Coord_Tuple>, Coord_Iterator_Tuple> IdItr;
    IdItr id_begin(pts->begin(), g->point_to_id);
    IdItr id_end(pts->end(), g->point_to_id);

    // Permutation from calculated point id to value in base vector
    typedef thrust::device_vector<int>::iterator IntItr;
    typedef thrust::permutation_iterator<IntItr, IdItr> BaseItr;
    BaseItr base_begin(g->base.begin(), id_begin);

    // Exclusive scan by key to get count for number of points in cell
    thrust::constant_iterator<int> one(1);
    thrust::exclusive_scan_by_key(g->cells.begin(), g->cells.end(), one, g->cells.begin());

    // 'Undo' previous sort to have an increasing point count per cell
    //thrust::stable_sort_by_key(cell_indices.begin(), cell_indices.end(), g->cells.begin());
    thrust::sort_by_key(cell_indices.begin(), cell_indices.end(), g->cells.begin());

    // Offset calculated base indices from permutation iterator by point per cell count
    thrust::transform(g->cells.begin(), g->cells.end(), base_begin, cell_indices.begin(), thrust::plus<int>());

    // Fill cells with increasing count
    thrust::sequence(g->cells.begin(), g->cells.end());

    // Reorder indices by offset base indices
    thrust::stable_sort_by_key(cell_indices.begin(), cell_indices.end(), g->cells.begin());
    //thrust::sort_by_key(cell_indices.begin(), cell_indices.end(), g->cells.begin());
    
    #ifdef DEBUG
    cout << "Cells: [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    #ifdef STATS
    g->Num_Points_Per_Cell.resize(g->ng3, 0);
    g->Num_Cells_Searched.resize(g->Max_Cells_Searched+1, 0);
    g->Num_Points_Checked.resize(g->Max_Points_Checked+1, 0);

    g->Min_Points_Per_Cell = -1;
    g->Max_Points_Per_Cell = -1;
    g->Avg_Points_Per_Cell = -1;
    g->Num_Fast_Queries = 0;
    g->Num_Slow_Queries = 0;
    g->Num_Exhaustive_Queries = 0;
    g->Total_Cells_Searched = 0;
    g->Total_Points_Checked = 0;
    g->Points_Checked = 0;

    thrust::adjacent_difference(g->base.begin(), g->base.end(), g->Num_Points_Per_Cell.begin());
    g->Min_Points_Per_Cell = *thrust::min_element(g->Num_Points_Per_Cell.begin()+1,
                                                  g->Num_Points_Per_Cell.end());
    g->Max_Points_Per_Cell = *thrust::max_element(g->Num_Points_Per_Cell.begin()+1,
                                                  g->Num_Points_Per_Cell.end());
    g->Avg_Points_Per_Cell = static_cast<float>(nfixpts) / static_cast<float>(g->ng3);
    #endif

    return g;
  }

  // Perform a single query
  template<typename Coord_T>
  void Query(Grid_T<Coord_T>* g, thrust::tuple<Coord_T, Coord_T, Coord_T>& q, int& closest) {
    // Get id of cell containing query
    const int queryint(g->point_to_id(q));
    // Get number of points in cell
    const int num_points_in_cell(g->num_points_in_cell_id(queryint));

    // If cell contains any points, perform a fast query
    if (num_points_in_cell > 0) {
      closest = g->fast_query(q);
    }
    else {
      // Perform a slow query
      closest = g->slow_query(q);
      // If query failed do exhaustive search
      if (closest < 0) {
        closest = g->exhaustive_query(q);
      }
    }
  }
  

  // Parallel query on preprocessed grid
  template<typename Coord_T>
  void Query(Grid_T<Coord_T>* g, Point_Vector<Coord_T>* q, thrust::host_vector<int>* closest) {
    // Typedefs derived from Grid class
    typedef typename Grid_T<Coord_T>::Coord_Tuple Coord_Tuple;
    typedef typename Grid_T<Coord_T>::Coord_Iterator_Tuple Coord_Iterator_Tuple;
    
    // Initialize vector of indices
    const int nqpts(q->get_size());
    thrust::device_vector<int> qindices(nqpts);
    thrust::sequence(qindices.begin(), qindices.end());
    thrust::device_vector<int> qcells(nqpts, -1);
    
    // Calculate cell id for query points
    thrust::transform(q->begin(), q->end(), qcells.begin(), g->point_to_id);

    #ifdef DEBUG
    cout << "Query IDs: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    // Calculate number of points in each query point's cell
    thrust::transform(qcells.begin(), qcells.end(), qcells.begin(), g->num_points_in_cell_id);
    
    #ifdef DEBUG
    cout << "Number of points in cells: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    // Zip iterator to reorder the cells and indices at same time
    typedef thrust::device_vector<int>::iterator IntItr;
    typedef thrust::zip_iterator<thrust::tuple<IntItr, IntItr> > ZipItr;
    greater_functor<thrust::tuple<int, int> > greater_zero(0);
    ZipItr index_begin(thrust::make_zip_iterator(thrust::make_tuple(qcells.begin(), qindices.begin())));
    ZipItr index_end(thrust::make_zip_iterator(thrust::make_tuple(qcells.end(), qindices.end())));
    // Partition by number of points in a cell, so all non-empty ones are together and can be iterated over
    ZipItr index_split = thrust::partition(index_begin, index_end, greater_zero);
    // Index of where the partition was split
    int split = index_split - index_begin;

    #ifdef DEBUG
    cout << "Partition: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    cout << "Indices: [";
    thrust::copy(qindices.begin(), qindices.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    // Permutation from indices to actual points
    typedef thrust::permutation_iterator<Coord_Iterator_Tuple, IntItr> QueryItr;
    QueryItr qbegin(q->begin(), qindices.begin());
    // Do fast case query on all query points that have points in the cell
    thrust::transform(qbegin, qbegin + split, qcells.begin(), g->fast_query);
    #ifdef STATS
    g->Num_Fast_Queries = split;
    #endif

    #ifdef DEBUG
    cout << "Fast on (" << 0 << ", " << split << ")" << endl;
    cout << "Fast Query Results: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    // Slow case query on empty cell queries
    thrust::transform(qbegin + split, qbegin + nqpts, qcells.begin() + split, g->slow_query);
    #ifdef STATS
    g->Num_Slow_Queries = nqpts - split;
    #endif

    #ifdef DEBUG
    cout << "Slow on (" << split << ", " << nqpts << ")" << endl;
    cout << "Slow Query Results: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    // Any slow case queries that returned -1 need to be done exhaustively.
    // It will be faster to do parallel search over all points, rather than parallel exhaustive searches
    greater_functor<thrust::tuple<int, int> > positive(-1);
    index_split = thrust::partition(index_begin, index_end, positive);
    split = index_split - index_begin;

    #ifdef DEBUG
    cout << "Repartition: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    // Perform exhaustive queries
    for (int i = split; i < nqpts; ++i) {
      qcells[i] = g->exhaustive_query((*q)[i]);
    }
    
    #ifdef STATS
    g->Num_Exhaustive_Queries = nqpts - split;
    #endif

    #ifdef DEBUG
    cout << "Exhaustive on (" << split << ", " << nqpts << ")" << endl;
    cout << "Exhaustive Query Results: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    thrust::sort_by_key(qindices.begin(), qindices.end(), qcells.begin());

    #ifdef DEBUG
    cout << "Resorted Query results: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    // Copy back to host
    thrust::copy(qcells.begin(), qcells.end(), closest->begin());
  }
};
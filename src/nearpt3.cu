#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/copy.h>

#include <algorithm>
#include <boost/multi_array.hpp> 
#include <iomanip>
#include <iostream>
#include <fstream>
#include <math.h>
#include <sys/resource.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <sys/times.h>
#include <sys/types.h>
#include <time.h>
#include <unistd.h>

#include "grid.cu"

using namespace std;
using boost::array;

namespace nearpt3 {
	double ng_factor = 1.6;

  // cellsearchorder:
  // First 3 elements of each row:  the order in which to search (one 48th-ant of the) cells adjacent to the current cell.
  // 4th element:   where, in cellsearchorder, to stop searching after the first point is found.
  const static int  cellsearchorder[][4] = {
#include "cellsearchorder"
  };
  // Number of cells in cellsearchorder (before expanding symmetries).
  const static int ncellsearchorder = 
    sizeof(nearpt3::cellsearchorder) / sizeof(nearpt3::cellsearchorder[0][0])/4;

  template<typename Coord_T> Grid_T<Coord_T>*
  Preprocess(const int nfixpts, Points_Vector<Coord_T>* pts) {
    typedef typename Grid_T<Coord_T>::Coord_Tuple Coord_Tuple;
    typedef typename Grid_T<Coord_T>::Coord_Iterator_Tuple Coord_Iterator_Tuple;
    
    Grid_T<Coord_T> *g;
    g = new Grid_T<Coord_T>;
    g->nfixpts = nfixpts;
    int &ng = g->ng;
    ng = static_cast<int> (ng_factor * cbrt(static_cast<double>(nfixpts)));

    ng = min(2000, max(1, ng));
    g->ng3 = ng * ng * ng;
    g->pts = pts;

    for (int i=1; i<ncellsearchorder; ++i)
      if (nearpt3::cellsearchorder[i-1][3] > nearpt3::cellsearchorder[i][3]) 
        throw "cellsearchorder is not monotonic";

    thrust::pair<array<Coord_T,3>, array<Coord_T,3> > minmax = pts->minmax();
    array<Coord_T,3> lo = thrust::get<0>(minmax);
    array<Coord_T,3> hi = thrust::get<1>(minmax);

    #ifdef DEBUG
    cout << "Min/Max" << endl;
    cout << lo[0] << ", " << lo[1] << ", " << lo[2] << endl;
    cout << hi[0] << ", " << hi[1] << ", " << hi[2] << endl;
    #endif

    array<double,3> s;
    for (int i=0; i<3; ++i) {
      s[i] = 0.99 * ng / static_cast<double>(hi[i] - lo[i]);
    }
    g->r_cell = min(min(s[0], s[1]), s[2]);

    for(int i=0; i<3; i++) {
      g->d_cell[i] = ((ng-1)-(lo[i]+hi[i])*g->r_cell) * 0.5;
    }

    point_to_id_functor<Coord_Tuple> g_point_to_id(g->ng, g->r_cell, g->d_cell[0],
                                                   g->d_cell[1], g->d_cell[2]);

    #ifdef DEBUG
    cout << "Grid info:";
    cout << "\nng: " << g->ng;
    cout << "\nng3: " << g->ng3;
    cout << "\ns: (" << s[0] << ", " << s[1] << ", " << s[2] << ")";
    cout << "\nr_cell: " << g->r_cell;
    cout << "\nd_cell: " << g->d_cell[0] << ", " << g->d_cell[1] << ", " << g->d_cell[2] << ")";
    cout << endl;
    #endif

    g->base = thrust::device_vector<int>(g->ng3+1, 1);
    g->cells = thrust::device_vector<int>(g->nfixpts);
    g->cell_indices = thrust::device_vector<int>(g->nfixpts);
    thrust::sequence(g->cell_indices.begin(), g->cell_indices.end());
    
    // Calculate cell id from point
    thrust::transform(pts->begin(), pts->end(), g->cells.begin(), g_point_to_id);

    #ifdef DEBUG
    cout << "Cell IDs (cells): [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    // Ensure no cells are -1 (outside range)
    if (thrust::find(g->cells.begin(), g->cells.end(), -1) != g->cells.end()) {
      throw "Bad cell";
    }

    // Keep track of the indices of cells during sorting
    thrust::stable_sort_by_key(g->cells.begin(), g->cells.end(), g->cell_indices.begin());

    #ifdef DEBUG
    cout << "Sorted Cell IDs (cells): [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    // Taken from thrust histogram example
    thrust::counting_iterator<int> count(0);
    thrust::lower_bound(g->cells.begin(), g->cells.end(),
                        count, count + g->ng3 + 1,
                        g->base.begin());

    #ifdef DEBUG
    cout << "Count: " << *count << endl;
    cout << "Lower bound (base): [";
    thrust::copy(g->base.begin(), g->base.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    if (g->base[g->ng3] != nfixpts) {
      cout << "ERROR: Internal inconsistency; wrong " << PRINTN(g->base[g->ng3]);
      throw "Internal inconsistency";
    }

    // Transform iterator to compute point ids then permutation iterator to get value from base
    typedef thrust::transform_iterator<point_to_id_functor<Coord_Tuple>, Coord_Iterator_Tuple> ptid_itr;
    ptid_itr ptid_begin(pts->begin(), g_point_to_id);
    ptid_itr ptid_end(pts->end(), g_point_to_id);

    typedef thrust::device_vector<int>::iterator IntItr;
    typedef thrust::permutation_iterator<IntItr, ptid_itr> PermItr;
    PermItr cbbegin(g->base.begin(), ptid_begin);

    // Exclusive scan by key to get count for number of points in cell
    thrust::constant_iterator<int> one(1);
    thrust::exclusive_scan_by_key(g->cells.begin(), g->cells.end(), one, g->cells.begin());

    // 'Undo' previous sort to have an increasing point count per cell
    thrust::stable_sort_by_key(g->cell_indices.begin(), g->cell_indices.end(), g->cells.begin());

    // Offset calculated base indices from permutation iterator by point per cell count
    thrust::plus<int> plus_op;
    thrust::transform(g->cells.begin(), g->cells.end(), cbbegin, g->cell_indices.begin(), plus_op);

    // Fill cells with increasing count
    thrust::sequence(g->cells.begin(), g->cells.end());

    // Reorder indices by offset base indices
    thrust::stable_sort_by_key(g->cell_indices.begin(), g->cell_indices.end(), g->cells.begin());
    
    #ifdef DEBUG
    cout << "Cell indices: [";
    thrust::copy(g->cell_indices.begin(), g->cell_indices.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    cout << "ID (cells iterator): [";
    thrust::copy(ptid_begin, ptid_end, ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    cout << "Permutation (cells): [";
    thrust::copy(cbbegin, cbbegin + g->nfixpts, ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    cout << "Point to cell (cells): [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    return g;
  }

  template<typename Coord_T> int
  Query(Grid_T<Coord_T>* g, const array<Coord_T, 3> q) {
    typedef typename Grid_T<Coord_T>::Coord_Tuple Coord_Tuple;
    typedef typename Grid_T<Coord_T>::Coord_Iterator_Tuple Coord_Iterator_Tuple;

    int closestpt(g->Query_Fast_Case(q));
    if (closestpt>=0) {
      return closestpt;
    }
    
    Cell3 querycell(g->Compute_Cell_Containing_Point(q));

    double dist(numeric_limits<double>::max());
    //int closecell(-1);
    //int goodsortnum;
    bool foundit(false);
    int nstop(ncellsearchorder);
    
    for (int isort=0; isort<nstop; ++isort) {
      int thisclosest;
      double thisdist;
      Cell3 s (cellsearchorder[isort][0], cellsearchorder[isort][1], 
	       cellsearchorder[isort][2]);

      for (int isign=0; isign<8; isign++) {      // Iterate over all combinations of signs;
        static const int sign3[8][3] = {{1,1,1},{1,1,-1},{1,-1,1},{1,-1,-1},
                                        {-1,1,1},{-1,1,-1},{-1,-1,1},{-1,-1,-1}};
        if (s[0]==0 && sign3[isign][0]== -1) continue;
        if (s[1]==0 && sign3[isign][1]== -1) continue;
        if (s[2]==0 && sign3[isign][2]== -1) continue;

        const Cell3 s2(s*sign3[isign]);

        for (int iperm=0; iperm<6; iperm++) {   // Iterate over all permutations of coordinates.
          switch (iperm) {
          case 1:
            if (s[1]==s[2]) continue;
            break;
          case 2: 
            if (s[0]==s[1]) continue;
            break;
          case 3:
          case 4:
            if (s[0]==s[1] && s[0]==s[2]) continue;
            break;
          case 5:
            if (s[0]==s[2]) continue;
            break;
          }
          static const int perm3[6][3] = {{0,1,2},{0,2,1},{1,0,2},{1,2,0},{2,0,1},{2,1,0}};
          const Cell3 s3(s2[perm3[iperm][0]], s2[perm3[iperm][1]], s2[perm3[iperm][2]]);
          const Cell3 c2(querycell+s3);
          if (!g->check(c2)) continue;  // outside the universe?
          //goodsortnum = isort;
          g->querythiscell(c2, q, thisclosest, thisdist);
          if (thisclosest < 0) continue;

          // If two fixed points are the same distance from the query, then return the one with the
          // smallest index.  This removes ambiguities, but complicates the code in several places.
          
          if (thisdist<dist || (thisdist==dist && thisclosest<closestpt)) {
            dist = thisdist;
            closestpt = thisclosest;
            //closecell = g->cellid_to_int(c2);
            if (!foundit) {
              foundit = true;
              nstop = cellsearchorder[isort][3];
              if (nstop >= ncellsearchorder) {
                // It took so long to find any cell with a point that cellsearchorder doesn't have
                // enough cells to be sure of finding the closest point.  Fall back to naive
                // exhaustive searching.
                goto L_end_isort;
              }
            }
          }
        }
      }
    }

  L_end_isort: if (closestpt>=0) {
      return closestpt;
    }
    
    // No nearby points, so exhaustively search over all the fixed points.
    typedef thrust::device_vector<int>::iterator IntItr;
    typedef thrust::transform_iterator<distance2_functor<Coord_Tuple>, Coord_Iterator_Tuple> dist2_itr;
    distance2_functor<Coord_Tuple> distance2(q[0], q[1], q[2]);
    dist2_itr begin(g->pts->begin(), distance2);
    dist2_itr end(g->pts->end(), distance2);
    dist2_itr result = thrust::min_element(begin, end);
    closestpt = g->cells[result - begin];
    
    return closestpt;
  }
  
};
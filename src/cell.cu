#pragma once
#include <hip/hip_runtime.h>


#include <iostream>

namespace nearpt3 {

  typedef short int Cell_Index_T;
  class Cell3 {
  public:
    Cell_Index_T c[3];

    __host__ __device__
    Cell3(const Cell_Index_T x, const Cell_Index_T y, const Cell_Index_T z) {
      c[0] = x; c[1] = y; c[2] = z; }

    __host__ __device__
    Cell3(const Cell3 &a) { c[0] = a[0]; c[1] = a[1]; c[2] = a[2]; }

    __host__ __device__
    Cell3() { c[0] = -1; c[1] = -1; c[2] = -1; }

    __host__ __device__
    Cell_Index_T & operator[] (const int i)  {  return c[i];  }

    __host__ __device__
    const Cell_Index_T & operator[] (const int i) const {  return c[i];  }

    __host__ __device__
    const Cell3 operator+(const Cell3 &d) const {
      Cell3 r;
      r[0] = c[0]+d[0];
      r[1] = c[1]+d[1];
      r[2] = c[2]+d[2];
      return r;
    }

    __host__ __device__
    const Cell3 operator*(const int *d) const {
      Cell3 r;
      r[0] = c[0]*d[0];
      r[1] = c[1]*d[1];
      r[2] = c[2]*d[2];
      return r;
    }

    __host__ __device__
    bool operator==(const Cell3 &d)  const {
      return c[0]==d[0] && c[1]==d[1] && c[2]==d[2];
    }

    //    const Cell3 operator*(const int *) const;
  };

  void write(std::ostream &o, const Cell3& c) {
    o << '(' << c[0] << ',' << c[1] << ',' << c[2] << ") ";
  }
};

#include "hip/hip_runtime.h"
#include <thrust/functional.h>
#include <limits>

#include "cell.cu"

namespace nearpt3 {

  
  // clamp_USI     Convert to an unsigned short int while clamping

  template <typename T> __host__ __device__
  unsigned short int clamp_USI(T a) {
    const T mm(static_cast<T>(USHRT_MAX));
    return  
      static_cast<unsigned short int>(a > mm ? mm : (a > 0 ? static_cast<unsigned short int>(a) : 0));
  }

  // Check if this is a legal cell.
  struct check_cell_functor
  {
    int ng;

    check_cell_functor() : ng(-1) {}
    check_cell_functor(int ng) : ng(ng) {}

    __host__ __device__
    bool operator()(const Cell3& a) const {
      if (a[0] < 0 || a[0] >= ng ) return false;
      if (a[1] < 0 || a[1] >= ng ) return false;
      if (a[2] < 0 || a[2] >= ng ) return false;
      return true;
    }
  };

  // clip is needed because roundoff errors may cause a number to be slightly outside the legal range
  struct clip_cell_functor
  {
    int ng;

    clip_cell_functor() : ng(-1) {}
    clip_cell_functor(int ng) : ng(ng) {}

    __host__ __device__
    void operator()(Cell3& a) {
      if (a[0] < 0) a[0] = 0;
      if (a[0] >= ng) a[0] = ng-1;
      if (a[1] < 0) a[1] = 0;
      if (a[1] >= ng) a[1] = ng-1;
      if (a[2] < 0) a[2] = 0;
      if (a[2] >= ng) a[2] = ng-1;
    }
  };

  template<typename Coord_Tuple>
  struct cell_containing_point_functor : public thrust::unary_function<Coord_Tuple, Cell3>
  {
    double r_cell;
    double d0;
    double d1;
    double d2;

    cell_containing_point_functor() : r_cell(-1), d0(-1), d1(-1), d2(-1) {}

    cell_containing_point_functor(double r_cell, double d0, double d1, double d2)
      : r_cell(r_cell), d0(d0), d1(d1), d2(d2) {}

    __host__ __device__
    Cell3 operator()(const Coord_Tuple& a) const {
      int ix = static_cast<short int>(static_cast<double>(thrust::get<0>(a))*r_cell+d0);
      int iy = static_cast<short int>(static_cast<double>(thrust::get<1>(a))*r_cell+d1);
      int iz = static_cast<short int>(static_cast<double>(thrust::get<2>(a))*r_cell+d2);
      Cell3 c(ix, iy, iz);
      return c;
    }
  };

  struct cell_to_id_functor : public thrust::unary_function<Cell3, int>
  {
    int ng;

    cell_to_id_functor() : ng(-1) {}
    cell_to_id_functor(int ng) : ng(ng) {}
  
    __host__ __device__
    int operator()(const Cell3& c) const {
      if (c[0]<0 || c[0] >=ng || c[1]<0 || c[1] >=ng || c[2]<0 || c[2] >=ng) return -1;
      return (static_cast<int> (c[0])*ng + static_cast<int>(c[1]))*ng + c[2];
    }
  };
  
  template<typename Coord_Tuple>
  struct point_to_id_functor : public thrust::unary_function<Coord_Tuple, int>
  {
    cell_containing_point_functor<Coord_Tuple> cell_containing_point;
    cell_to_id_functor cell_to_id;

    point_to_id_functor() : cell_containing_point(cell_containing_point_functor<Coord_Tuple>()),
                            cell_to_id(cell_to_id_functor()) {}

    point_to_id_functor(cell_containing_point_functor<Coord_Tuple> cell_containing_point,
                        cell_to_id_functor cell_to_id)
      : cell_containing_point(cell_containing_point), cell_to_id(cell_to_id) {}

    __host__ __device__
    int operator()(const Coord_Tuple& a) const {
      return cell_to_id(cell_containing_point(a));
    }
  };

  struct num_points_in_cell_id_functor : public thrust::unary_function<int, int>
  {
    thrust::device_ptr<int> base;

    num_points_in_cell_id_functor() : base(thrust::device_ptr<int>()) {}
    num_points_in_cell_id_functor(thrust::device_ptr<int> base) : base(base) {}

    __host__ __device__
    int operator()(const int& id) const {
      if (id < 0) return 0;
      return base[id+1] - base[id];
    }
  };

  template<typename Coord_Tuple>
  struct distance2_functor : public thrust::unary_function<Coord_Tuple, double>
  {
    const Coord_Tuple q;

    __host__ __device__
    distance2_functor(Coord_Tuple q) : q(q) { }

    __host__ __device__
    double square(const double t) const {
      return t * t;
    }
  
    __host__ __device__
    double operator()(const Coord_Tuple& a) const {
      return (square((thrust::get<0>(a)-thrust::get<0>(q))) +
              square((thrust::get<1>(a)-thrust::get<1>(q))) +
              square((thrust::get<2>(a)-thrust::get<2>(q))));
    }
  };

  template<typename Coord_T>
  struct query_cell_functor
  {
    typedef thrust::tuple<Coord_T, Coord_T, Coord_T> Coord_Tuple;
    typedef thrust::device_ptr<Coord_T> Coord_Ptr;
    typedef thrust::tuple<Coord_Ptr, Coord_Ptr, Coord_Ptr> Coord_Ptr_Tuple;
    
    num_points_in_cell_id_functor num_points_in_cell_id;
    point_to_id_functor<Coord_Tuple> point_to_id;
    thrust::device_ptr<int> base;
    thrust::device_ptr<int> cells;
    Coord_Ptr_Tuple pts;

    query_cell_functor() : num_points_in_cell_id(num_points_in_cell_id_functor()),
                           point_to_id(point_to_id_functor<Coord_Tuple>()),
                           base(thrust::device_ptr<int>()),
                           cells(thrust::device_ptr<int>()),
                           pts(thrust::make_tuple(Coord_Ptr(), Coord_Ptr(), Coord_Ptr())) {}
    
    query_cell_functor(num_points_in_cell_id_functor num_points_in_cell_id,
                       point_to_id_functor<Coord_Tuple> point_to_id,
                       thrust::device_ptr<int> base,
                       thrust::device_ptr<int> cells,
                       Coord_Ptr_Tuple pts)
      : num_points_in_cell_id(num_points_in_cell_id), point_to_id(point_to_id),
        base(base), cells(cells), pts(pts) {}

    __host__ __device__
    Coord_Tuple point_at(int i) {
      return thrust::make_tuple(thrust::get<0>(pts)[i],
                                thrust::get<1>(pts)[i],
                                thrust::get<2>(pts)[i]);
    }

    __host__ __device__
    void operator()(const int &cell_id, const Coord_Tuple& q,
                    int &closest, double &dist2) {
      const int num_points(num_points_in_cell_id(cell_id));
      if (num_points <= 0) {
        closest = -1;
        dist2 = -1;
        return;
      }
      
      const int queryint(point_to_id(q));
      distance2_functor<Coord_Tuple> distance2(q);
      int i = base[cell_id];
      closest = cells[i];
      dist2 = distance2(point_at(closest));
      while (i < base[cell_id+1]) {
        const double d2 = distance2(point_at(cells[i]));
        if (d2 < dist2 || (d2 == dist2 && cells[i] < closest)) {
          dist2 = d2;
          closest = cells[i];
        }
        ++i;
      }
    }
  };

  template<typename Coord_T>
  struct fast_query_functor : public thrust::unary_function<thrust::tuple<Coord_T, Coord_T, Coord_T>, int>
  {
    typedef thrust::tuple<Coord_T, Coord_T, Coord_T> Coord_Tuple;
    typedef thrust::device_ptr<Coord_T> Coord_Ptr;
    typedef thrust::tuple<Coord_Ptr, Coord_Ptr, Coord_Ptr> Coord_Ptr_Tuple;
    
    clip_cell_functor clip_cell;
    cell_containing_point_functor<Coord_Tuple> cell_containing_point;
    query_cell_functor<Coord_T> query_cell;

    fast_query_functor() : clip_cell(clip_cell_functor()),
                           cell_containing_point(cell_containing_point_functor<Coord_Tuple>()),
                           query_cell(query_cell_functor<Coord_T>()) {}

    fast_query_functor(clip_cell_functor clip_cell,
                       cell_containing_point_functor<Coord_Tuple> cell_containing_point,
                       query_cell_functor<Coord_T> query_cell)
      : clip_cell(clip_cell), cell_containing_point(cell_containing_point), query_cell(query_cell) {}

    __host__ __device__
    Coord_Tuple point_at(int i) {
      Coord_Ptr_Tuple pts(query_cell.pts);
      return thrust::make_tuple(thrust::get<0>(pts)[i],
                                thrust::get<1>(pts)[i],
                                thrust::get<2>(pts)[i]);
    }
    
    __host__ __device__
    int operator()(const Coord_Tuple& q) {
      int queryint = query_cell.point_to_id(q);
      int closestpt = -1;
      double dist2 = -1;
      query_cell(queryint, q, closestpt, dist2);
      const double distf = sqrt(dist2) * 1.00001;
      Coord_Tuple lopt(thrust::make_tuple(clamp_USI(static_cast<double>(thrust::get<0>(q)) - distf),
                                          clamp_USI(static_cast<double>(thrust::get<1>(q)) - distf),
                                          clamp_USI(static_cast<double>(thrust::get<2>(q)) - distf)));
      Coord_Tuple hipt(thrust::make_tuple(clamp_USI(static_cast<double>(thrust::get<0>(q)) + distf + 1.0),
                                          clamp_USI(static_cast<double>(thrust::get<1>(q)) + distf + 1.0),
                                          clamp_USI(static_cast<double>(thrust::get<2>(q)) + distf + 1.0)));
      
      Cell3 locell(cell_containing_point(lopt));
      Cell3 hicell(cell_containing_point(hipt));

      clip_cell(locell);
      clip_cell(hicell);

      Cell3 qcell(cell_containing_point(q));
      if (locell == qcell && hicell == qcell) {
        return closestpt;
      }
      int close2 = -1;
      double d2 = -1;
      for (Coord_T x=locell[0]; x<=hicell[0]; ++x) {
        for (Coord_T y=locell[1]; y<=hicell[1]; ++y) {
          for (Coord_T z=locell[2]; z<=hicell[2]; ++z) {
            queryint = query_cell.point_to_id.cell_to_id(Cell3(x, y, z));
            query_cell(queryint, q, close2, d2);
            if (close2 != -1 && (d2 < dist2 || (d2 == dist2 && close2 < closestpt))) {
              closestpt = close2;
              dist2 = d2;
            }
          }
        }
      }
      return closestpt;
    }
  };

  template<typename Coord_T>
  struct slow_query_functor : public thrust::unary_function<thrust::tuple<Coord_T, Coord_T, Coord_T>, int>
  {
    typedef thrust::tuple<Coord_T, Coord_T, Coord_T> Coord_Tuple;
    typedef thrust::device_ptr<Coord_T> Coord_Ptr;
    typedef thrust::tuple<Coord_Ptr, Coord_Ptr, Coord_Ptr> Coord_Ptr_Tuple;

    int ncellsearch;
    thrust::device_ptr<int> cellsearch;

    check_cell_functor check_cell;
    cell_containing_point_functor<Coord_Tuple> cell_containing_point;
    query_cell_functor<Coord_T> query_cell;

    slow_query_functor() : ncellsearch(0),
                           cellsearch(thrust::device_ptr<int>()),
                           check_cell(check_cell_functor()),
                           cell_containing_point(cell_containing_point_functor<Coord_Tuple>()),
                           query_cell(query_cell_functor<Coord_T>()) {}

    slow_query_functor(int ncellsearch,
                       thrust::device_ptr<int> cellsearch,
                       check_cell_functor check_cell,
                       cell_containing_point_functor<Coord_Tuple> cell_containing_point,
                       query_cell_functor<Coord_T> query_cell)
      : ncellsearch(ncellsearch), cellsearch(cellsearch), check_cell(check_cell),
        cell_containing_point(cell_containing_point), query_cell(query_cell) {}

    
    __host__ __device__
    Coord_Tuple point_at(int i) {
      Coord_Ptr_Tuple pts(query_cell.pts);
      return thrust::make_tuple(thrust::get<0>(pts)[i],
                                thrust::get<1>(pts)[i],
                                thrust::get<2>(pts)[i]);
    }
    
    __host__ __device__
    int operator()(const Coord_Tuple& q) {
      const int sign3[8][3] = {{1,1,1},{1,1,-1},{1,-1,1},{1,-1,-1},
                               {-1,1,1},{-1,1,-1},{-1,-1,1},{-1,-1,-1}};
      const int perm3[6][3] = {{0,1,2},{0,2,1},{1,0,2},{1,2,0},{2,0,1},{2,1,0}};

      Cell3 qcell = cell_containing_point(q);
      int queryint = query_cell.point_to_id.cell_to_id(qcell);
      int closestpt = -1;
      double dist2 = -1;

      int nstop(ncellsearch);
      bool found(false);

      for (int isort=0; isort<nstop; ++isort) {
        int close2;
        double d2;
        Cell3 s (cellsearch[isort*4], cellsearch[isort*4+1], cellsearch[isort*4+2]);

        for (int isign=0; isign<8; ++isign) {
          if (s[0]==0 && sign3[isign][0]== -1) continue;
          if (s[1]==0 && sign3[isign][1]== -1) continue;
          if (s[2]==0 && sign3[isign][2]== -1) continue;

          const Cell3 s2(s*sign3[isign]);

          for (int iperm=0; iperm<6; ++iperm) {
            switch (iperm) {
            case 1:
              if (s[1]==s[2]) continue;
              break;
            case 2: 
              if (s[0]==s[1]) continue;
              break;
            case 3:
            case 4:
              if (s[0]==s[1] && s[0]==s[2]) continue;
              break;
            case 5:
              if (s[0]==s[2]) continue;
              break;
            }
            const Cell3 s3(s2[perm3[iperm][0]], s2[perm3[iperm][1]], s2[perm3[iperm][2]]);
            const Cell3 c2(qcell+s3);
            if (!check_cell(c2)) continue;
            int cell_id(query_cell.point_to_id.cell_to_id(c2));
            query_cell(cell_id, q, close2, d2);
            if (close2 < 0) continue;

            if (dist2 == -1 || d2 < dist2 || (d2 == dist2 && close2 < closestpt)) {
              dist2 = d2;
              closestpt = close2;
              if (!found) {
                found = true;
                nstop = cellsearch[isort*4+3];
                if (nstop >= ncellsearch) {
                  iperm = 6;
                  isign = 8;
                  isort = nstop;
                }
              }
            }
          }
        }
      }
      return closestpt;
    }
  };
  

  template<typename T>
  struct greater_functor : public thrust::unary_function<T, bool>
  {
    const int b;

    greater_functor(int b) : b(b) {}

    __host__ __device__
    bool operator()(const T& a) const {
      return thrust::get<0>(a) > b;
    }
  };
};
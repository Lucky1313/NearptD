#include "hip/hip_runtime.h"
#pragma once

#ifdef DEBUG
#define PROFILE
#endif

#include <iostream>

#include <thrust/pair.h>
#include <thrust/tuple.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#ifdef STATS
#include <thrust/adjacent_difference.h>
#endif

#include "point_vector.cu"
#include "functors.cu"
#include "cellsearchorder.cu"
#include "grid.cu"
#include "utils.cpp"
#include "timer.cpp"
#include "tuple_utility.cu"

using namespace std;

namespace nearptd {
  
	double ng_factor = 1.6;

  // Process all fixed points into a uniform grid
  template<typename Coord_T, size_t Dim>
  Grid_T<Coord_T, Dim>*
  Preprocess(const int nfixpts, Point_Vector<Coord_T, Dim>* pts) {
    // Typedefs derived from Grid class
    typedef typename Grid_T<Coord_T, Dim>::Coord_Tuple Coord_Tuple;
    typedef typename Grid_T<Coord_T, Dim>::Coord_Tuple_Iterator Coord_Tuple_Iterator;
    typedef typename Grid_T<Coord_T, Dim>::Double_Tuple Double_Tuple;

    #ifdef PROFILE
    Timer pptimer = Timer(true);
    #endif
    
    Grid_T<Coord_T, Dim> *g;
    g = new Grid_T<Coord_T, Dim>;
    
    g->nfixpts = nfixpts;
    int &ng = g->ng;
    // Multiply by the dim root of the number of points
    ng = static_cast<int> (ng_factor * pow(static_cast<double>(nfixpts), 1.0 / static_cast<double>(Dim)));

    ng = min(2000, max(1, ng));
    g->ngd = pow(ng, Dim);
    g->pts = pts;

    // Get min and max of data range
    thrust::pair<Coord_Tuple, Coord_Tuple> minmax = pts->minmax();
    Coord_Tuple lo = thrust::get<0>(minmax);
    Coord_Tuple hi = thrust::get<1>(minmax);

    // Calculate scale for each dimension
    scale<Coord_T> sc(ng);
    tuple_binary_apply<Coord_Tuple, Coord_Tuple, Double_Tuple, scale<Coord_T>, Dim> make_scale;
    Double_Tuple s = make_scale(lo, hi, sc);

    // Find minimum scale of all dimensions
    thrust::minimum<double> min;
    tuple_reduce<Double_Tuple, double, thrust::minimum<double>, Dim> minimum;
    g->r_cell = minimum(s, min);
    // Calculate size of cell for each dimension
    cell_dim<Coord_T> cd(ng, g->r_cell);
    tuple_binary_apply<Coord_Tuple, Coord_Tuple, Double_Tuple, cell_dim<Coord_T>, Dim> make_cell_dim;
    g->d_cell = make_cell_dim(lo, hi, cd);

    #ifdef PROFILE
    pptimer("Create grid parameters");
    #endif
    
    // Create device vectors (Must be before functors)
    g->base = thrust::device_vector<int>(g->ngd+1, 1);
    g->cells = thrust::device_vector<int>(g->nfixpts);

    // Create cell search order
    g->cellsearch = cellsearchorder<Dim>();

    #ifdef PROFILE
    pptimer("Create cell search order");
    #endif
    
    // Define grid functors
    g->check_cell = check_cell_functor<Dim>(g->ng);
    g->clip_cell = clip_cell_functor<Dim>(g->ng);
    g->cell_containing_point = cell_containing_point_functor<Coord_T, Dim>(g->r_cell, g->d_cell);
    g->cell_to_id = cell_to_id_functor<Dim>(g->ng);
    g->point_to_id = point_to_id_functor<Coord_T, Dim>(g->cell_containing_point, g->cell_to_id);
    g->num_points_in_cell_id = num_points_in_cell_id_functor(g->base.data());
    g->query_cell = query_cell_functor<Coord_T, Dim>(g->num_points_in_cell_id,
                                                     g->point_to_id,
                                                     g->base.data(),
                                                     g->cells.data(),
                                                     pts->get_ptrs());
    g->fast_query = fast_query_functor<Coord_T, Dim>(g->clip_cell,
                                                     g->cell_containing_point,
                                                     g->query_cell);
    g->slow_query = slow_query_functor<Coord_T, Dim>(g->cellsearch.size,
                                                     g->cellsearch.cells->get_ptrs(),
                                                     g->cellsearch.stop.data(),
                                                     g->check_cell,
                                                     g->cell_containing_point,
                                                     g->query_cell);
    
    #ifdef DEBUG
    cout << "Grid info:";
    cout << "\nng: " << g->ng;
    cout << "\nngd: " << g->ngd;
    cout << "\nr_cell: " << g->r_cell;
    cout << endl;
    #endif

    #ifdef PROFILE
    pptimer("Create grid functors");
    #endif

    // Create index of cells to reorder cells later
    thrust::device_vector<int> cell_indices(g->nfixpts);
    thrust::sequence(cell_indices.begin(), cell_indices.end());
    
    // Calculate cell id from point
    thrust::transform(pts->begin(), pts->end(), g->cells.begin(), g->point_to_id);
    
    #ifdef PROFILE
    pptimer("Calculate point ids");
    #endif

    // Ensure no cells are -1 (outside range)
    if (thrust::find(g->cells.begin(), g->cells.end(), -1) != g->cells.end()) {
      throw "Bad cell";
    }

    // Sort by ids of cells
    thrust::stable_sort_by_key(g->cells.begin(), g->cells.end(), cell_indices.begin());
    
    // Find where cells start and stop
    thrust::counting_iterator<int> count(0);
    thrust::lower_bound(g->cells.begin(), g->cells.end(),
                        count, count + g->ngd + 1,
                        g->base.begin());

    #ifdef DEBUG
    cout << "Base: [";
    thrust::copy(g->base.begin(), g->base.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    #ifdef PROFILE
    pptimer("Calculate base vector");
    #endif

    if (g->base[g->ngd] != nfixpts) {
      cout << "ERROR: Internal inconsistency; wrong point count: " << g->base[g->ngd];
      throw "Internal inconsistency";
    }

    // Transform iterator to compute point ids 
    typedef thrust::transform_iterator<point_to_id_functor<Coord_T, Dim>, Coord_Tuple_Iterator> IdItr;
    IdItr id_begin(pts->begin(), g->point_to_id);
    IdItr id_end(pts->end(), g->point_to_id);

    // Permutation from calculated point id to value in base vector
    typedef thrust::device_vector<int>::iterator IntItr;
    typedef thrust::permutation_iterator<IntItr, IdItr> BaseItr;
    BaseItr base_begin(g->base.begin(), id_begin);

    // Exclusive scan by key to get count for number of points in cell
    thrust::constant_iterator<int> one(1);
    thrust::exclusive_scan_by_key(g->cells.begin(), g->cells.end(), one, g->cells.begin());

    // 'Undo' previous sort to have an increasing point count per cell
    thrust::sort_by_key(cell_indices.begin(), cell_indices.end(), g->cells.begin());

    // Offset calculated base indices from permutation iterator by point per cell count
    thrust::transform(g->cells.begin(), g->cells.end(), base_begin, cell_indices.begin(), thrust::plus<int>());

    // Fill cells with increasing count
    thrust::sequence(g->cells.begin(), g->cells.end());

    // Reorder indices by offset base indices
    thrust::stable_sort_by_key(cell_indices.begin(), cell_indices.end(), g->cells.begin());
    
    #ifdef DEBUG
    cout << "Cells: [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    #ifdef PROFILE
    pptimer("Calculate cells vector");
    #endif

    #ifdef STATS
    g->Num_Points_Per_Cell.resize(g->ngd, 0);
    g->Num_Cells_Searched.resize(g->Max_Cells_Searched+1, 0);
    g->Num_Points_Checked.resize(g->Max_Points_Checked+1, 0);

    g->Min_Points_Per_Cell = -1;
    g->Max_Points_Per_Cell = -1;
    g->Avg_Points_Per_Cell = -1;
    g->Num_Fast_Queries = 0;
    g->Num_Slow_Queries = 0;
    g->Num_Exhaustive_Queries = 0;
    g->Total_Cells_Searched = 0;
    g->Total_Points_Checked = 0;
    g->Points_Checked = 0;

    thrust::adjacent_difference(g->base.begin(), g->base.end(), g->Num_Points_Per_Cell.begin());
    g->Min_Points_Per_Cell = *thrust::min_element(g->Num_Points_Per_Cell.begin()+1,
                                                  g->Num_Points_Per_Cell.end());
    g->Max_Points_Per_Cell = *thrust::max_element(g->Num_Points_Per_Cell.begin()+1,
                                                  g->Num_Points_Per_Cell.end());
    g->Avg_Points_Per_Cell = static_cast<float>(nfixpts) / static_cast<float>(g->ngd);
    #endif

    return g;
  }

  // Perform a single query on a coordinate tuple
  template<typename Coord_T, size_t Dim>
  void Query(Grid_T<Coord_T, Dim>* g, const typename ntuple<Coord_T, Dim>::tuple& q, int& closest) {
    // Get id of cell containing query
    const int queryint(g->point_to_id(q));
    // Get number of points in cell
    const int num_points_in_cell(g->num_points_in_cell_id(queryint));

    // If cell contains any points, perform a fast query
    if (num_points_in_cell > 0) {
      closest = g->fast_query(q);
    }
    else {
      // Perform a slow query
      closest = g->slow_query(q);
      // If query failed do exhaustive search
      if (closest < 0) {
        closest = g->exhaustive_query(q);
      }
    }
  }

  // Perform a single query on coordinate array, rather than tuple
  template<typename Coord_T, size_t Dim>
  void Query(Grid_T<Coord_T, Dim>* g, Coord_T* q, int& closest) {
    typedef typename Grid_T<Coord_T, Dim>::Coord_Tuple Coord_Tuple;
    ntuple<Coord_T, Dim> Coord_Ntuple;
    Coord_Tuple qt = Coord_Ntuple.make(q);
    Query(g, qt, closest);
  }
  

  // Perform parallel queries
  template<typename Coord_T, size_t Dim>
  void Query(Grid_T<Coord_T, Dim>* g, Point_Vector<Coord_T, Dim>* q, thrust::host_vector<int>* closest) {
    // Typedefs derived from Grid class
    typedef typename Grid_T<Coord_T, Dim>::Coord_Tuple Coord_Tuple;
    typedef typename Grid_T<Coord_T, Dim>::Coord_Tuple_Iterator Coord_Tuple_Iterator;

    #ifdef PROFILE
    Timer qtimer = Timer(true);
    #endif
    
    // Initialize vector of indices
    const int nqpts(q->get_size());
    thrust::device_vector<int> qindices(nqpts);
    thrust::sequence(qindices.begin(), qindices.end());
    thrust::device_vector<int> qcells(nqpts, -1);
    
    // Calculate cell id for query points
    thrust::transform(q->begin(), q->end(), qcells.begin(), g->point_to_id);

    #ifdef DEBUG
    cout << "Query IDs: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    #ifdef PROFILE
    qtimer("Calculate point ids");
    #endif
    
    // Calculate number of points in each query point's cell
    thrust::transform(qcells.begin(), qcells.end(), qcells.begin(), g->num_points_in_cell_id);
    
    #ifdef DEBUG
    cout << "Number of points in cells: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    #ifdef PROFILE
    qtimer("Calculate number of points in cell");
    #endif
    
    // Zip iterator to reorder the cells and indices at same time
    typedef thrust::device_vector<int>::iterator IntItr;
    typedef thrust::zip_iterator<thrust::tuple<IntItr, IntItr> > ZipItr;
    greater_functor<thrust::tuple<int, int> > greater_zero(0);
    ZipItr index_begin(thrust::make_zip_iterator(thrust::make_tuple(qcells.begin(), qindices.begin())));
    ZipItr index_end(thrust::make_zip_iterator(thrust::make_tuple(qcells.end(), qindices.end())));

    // Partition by number of points in a cell, so all non-empty ones are together and can be iterated over
    ZipItr index_split = thrust::partition(index_begin, index_end, greater_zero);
    // Index of where the partition was split
    int split = index_split - index_begin;

    #ifdef DEBUG
    cout << "Partition: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    cout << "Indices: [";
    thrust::copy(qindices.begin(), qindices.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    #ifdef PROFILE
    qtimer("Partition points");
    #endif

    // Permutation from indices to actual points
    typedef thrust::permutation_iterator<Coord_Tuple_Iterator, IntItr> QueryItr;
    QueryItr qbegin(q->begin(), qindices.begin());
    // Do fast case query on all query points that have points in the cell
    thrust::transform(qbegin, qbegin + split, qcells.begin(), g->fast_query);

    #ifdef STATS
    g->Num_Fast_Queries = split;
    #endif

    #ifdef DEBUG
    cout << "Fast on (" << 0 << ", " << split << ")" << endl;
    cout << "Fast Query Results: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    #ifdef PROFILE
    qtimer("Calculate fast queries");
    #endif

    // Slow case query on empty cell queries
    thrust::transform(qbegin + split, qbegin + nqpts, qcells.begin() + split, g->slow_query);

    #ifdef STATS
    g->Num_Slow_Queries = nqpts - split;
    #endif

    #ifdef DEBUG
    cout << "Slow on (" << split << ", " << nqpts << ")" << endl;
    cout << "Slow Query Results: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    #ifdef PROFILE
    qtimer("Calculate slow queries");
    #endif

    // Any slow case queries that returned -1 need to be done exhaustively.
    // Faster to do parallel search over all points, rather than parallel exhaustive searches
    greater_functor<thrust::tuple<int, int> > positive(-1);
    index_split = thrust::partition(index_begin, index_end, positive);
    split = index_split - index_begin;

    #ifdef DEBUG
    cout << "Repartition: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    #ifdef PROFILE
    qtimer("Repartition");
    #endif

    // Perform exhaustive queries
    for (int i = split; i < nqpts; ++i) {
      qcells[i] = g->exhaustive_query((*q)[i]);
    }
    
    #ifdef STATS
    g->Num_Exhaustive_Queries = nqpts - split;
    #endif

    #ifdef DEBUG
    cout << "Exhaustive on (" << split << ", " << nqpts << ")" << endl;
    cout << "Exhaustive Query Results: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    #ifdef PROFILE
    qtimer("Calculate exhaustive queries");
    #endif

    // Resort queries back to given order
    thrust::sort_by_key(qindices.begin(), qindices.end(), qcells.begin());

    #ifdef DEBUG
    cout << "Resorted Query results: [";
    thrust::copy(qcells.begin(), qcells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    // Copy back to host
    thrust::copy(qcells.begin(), qcells.end(), closest->begin());
    
    #ifdef PROFILE
    qtimer("Sort and copy");
    #endif
  }
};
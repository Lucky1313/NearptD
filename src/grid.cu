#include "hip/hip_runtime.h"
#pragma once

#include <thrust/device_vector.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/extrema.h>

#include "point_vector.cu"
#include "functors.cu"
#include "tuple_utility.cu"

using namespace std;

namespace nearptd {

  // Check if this is a legal cell.
  template<size_t Dim>
  struct check_cell_functor
  {
    int ng;

    check_cell_functor() : ng(-1) {}
    check_cell_functor(int ng) : ng(ng) {}

    __host__ __device__
    bool operator()(const Cell<Dim>& a) const {
      for (int i=0; i<Dim; ++i) {
        if (a[i] < 0 || a[i] >= ng) return false;
      }
      return true;
    }
  };
  
  // clip is needed because roundoff errors may cause a number to be slightly outside the legal range
  template<size_t Dim>
  struct clip_cell_functor
  {
    int ng;

    clip_cell_functor() : ng(-1) {}
    clip_cell_functor(int ng) : ng(ng) {}

    __host__ __device__
    void operator()(Cell<Dim>& a) {
      for (size_t i=0; i<Dim; ++i) {
        if (a[i] < 0) a[i] = 0;
        if (a[i] >= ng) a[i] = ng-1;
      }
    }
  };

  // Return a cell that will contain a given point
  template<typename Coord_T, size_t Dim>
  struct cell_containing_point_functor
    : public thrust::unary_function<typename ntuple<Coord_T, Dim>::tuple, Cell<Dim> >
  {
    typedef typename Cell<Dim>::Cell_Index_T Cell_Index_T;
    typedef typename ntuple<double, Dim>::tuple Double_Tuple;
    typedef typename ntuple<Coord_T, Dim>::tuple Coord_Tuple;
    typedef typename ntuple<Cell_Index_T, Dim>::tuple Cell_Tuple;
    ntuple<double, Dim> Double_Ntuple;
    
    coord_to_cell_index<Coord_T, Cell_Index_T> cts;
    tuple_binary_apply<Coord_Tuple, Double_Tuple, Cell_Tuple,
      coord_to_cell_index<Coord_T, Cell_Index_T>, Dim> make_cell;

    double r_cell;
    Double_Tuple d_cell;
    
    cell_containing_point_functor() : r_cell(-1) {
      double d[Dim];
      for (int i=0; i<Dim; ++i) {d[i] = -1;}
      d_cell = Double_Ntuple.make_tuple(d);
    }

    cell_containing_point_functor(double r_cell, Double_Tuple d_cell)
      : r_cell(r_cell), d_cell(d_cell) {
      cts = coord_to_cell_index<Coord_T, Cell_Index_T>(r_cell);
    }

    __host__ __device__
    Cell<Dim> operator()(const Coord_Tuple& a) const {
      Cell<Dim> c(make_cell(a, d_cell, cts));
      return c;
    }
  };

  // Converts a cell to an id in a grid
  template<size_t Dim>
  struct cell_to_id_functor : public thrust::unary_function<Cell<Dim>, int>
  {
    int ng;

    cell_to_id_functor() : ng(-1) {}
    cell_to_id_functor(int ng) : ng(ng) {}
  
    __host__ __device__
    int operator()(const Cell<Dim>& c) const {
      int id = 0;
      for (int i=0; i<Dim; ++i) {
        if (c[i] < 0 || c[i] >= ng) return -1;
        id = static_cast<int>(c[i]) + ng * id;
      }
      return id;
    }
  };

  // Converts a given point to it's id in grid
  template<typename Coord_T, size_t Dim>
  struct point_to_id_functor : public thrust::unary_function<typename ntuple<Coord_T, Dim>::tuple, int>
  {
    typedef typename ntuple<Coord_T, Dim>::tuple Coord_Tuple;
    
    cell_containing_point_functor<Coord_T, Dim> cell_containing_point;
    cell_to_id_functor<Dim> cell_to_id;

    point_to_id_functor() : cell_containing_point(cell_containing_point_functor<Coord_T, Dim>()),
                            cell_to_id(cell_to_id_functor<Dim>()) {}
    
    point_to_id_functor(cell_containing_point_functor<Coord_T, Dim> cell_containing_point,
                        cell_to_id_functor<Dim> cell_to_id)
      : cell_containing_point(cell_containing_point), cell_to_id(cell_to_id) {}

    __host__ __device__
    int operator()(const Coord_Tuple& a) const {
      return cell_to_id(cell_containing_point(a));
    }
  };

  // Return number of points in a given cell
  struct num_points_in_cell_id_functor : public thrust::unary_function<int, int>
  {
    thrust::device_ptr<int> base;

    num_points_in_cell_id_functor() : base(thrust::device_ptr<int>()) {}
    num_points_in_cell_id_functor(thrust::device_ptr<int> base) : base(base) {}

    __host__ __device__
    int operator()(const int& id) const {
      if (id < 0) return 0;
      return base[id+1] - base[id];
    }
  };

  // Query a given cell, finding the minimum distance and index of closest point
  template<typename Coord_T, size_t Dim>
  struct query_cell_functor
  {
    // Convenience typedefs
    typedef typename ntuple<Coord_T, Dim>::tuple Coord_Tuple;
    typedef thrust::device_ptr<Coord_T> Coord_Ptr;
    typedef typename ntuple<Coord_Ptr, Dim>::tuple Coord_Ptr_Tuple;
    typedef typename ntuple<int, Dim>::tuple Int_Tuple;
    ntuple<Coord_Ptr, Dim> Coord_Ptr_Ntuple;
    ntuple<int, Dim> Int_Ntuple;

    // Tuple functors
    get_point<Coord_Ptr, Coord_T> point;
    tuple_binary_apply<Coord_Ptr_Tuple, Int_Tuple,
      Coord_Tuple, get_point<Coord_Ptr, Coord_T>, Dim> pget;

    // Grid functors
    num_points_in_cell_id_functor num_points_in_cell_id;
    point_to_id_functor<Coord_T, Dim> point_to_id;

    thrust::device_ptr<int> base;
    thrust::device_ptr<int> cells;
    Coord_Ptr_Tuple pts;

    query_cell_functor() : num_points_in_cell_id(num_points_in_cell_id_functor()),
                           point_to_id(point_to_id_functor<Coord_T, Dim>()),
                           base(thrust::device_ptr<int>()),
                           cells(thrust::device_ptr<int>()) {
      Coord_Ptr ptrs[Dim];
      for (size_t i=0; i<Dim; ++i) {ptrs[i] = Coord_Ptr();}
      pts = Coord_Ptr_Ntuple.make_tuple(ptrs);
    }
    
    query_cell_functor(num_points_in_cell_id_functor num_points_in_cell_id,
                       point_to_id_functor<Coord_T, Dim> point_to_id,
                       thrust::device_ptr<int> base,
                       thrust::device_ptr<int> cells,
                       Coord_Ptr_Tuple pts)
      : num_points_in_cell_id(num_points_in_cell_id), point_to_id(point_to_id),
        base(base), cells(cells), pts(pts) {}

    // Get coord tuple at point index
    __host__ __device__
    Coord_Tuple point_at(const int i) {
      int id[Dim];
      for (int d=0; d<Dim; ++d) {id[d] = i;}
      Int_Tuple id_tup = Int_Ntuple.make_tuple(id);
      return pget(pts, id_tup, point);
    }
    
    __host__ __device__
    void operator()(const int &cell_id, const Coord_Tuple& q,
                    int &closest, double &dist2) {
      const int num_points(num_points_in_cell_id(cell_id));
      // Empty cell
      if (num_points <= 0) {
        closest = -1;
        dist2 = -1;
        return;
      }
      
      const int queryint(point_to_id(q));
      distance2_functor<Coord_T, Dim> distance2(q);
      // Start with minimum as first point in cell
      closest = cells[base[cell_id]];
      dist2 = distance2(point_at(closest));
      // Loop through points in cell
      for (int i = base[cell_id]; i < base[cell_id+1]; ++i) {
        const double d2 = distance2(point_at(cells[i]));
        if (d2 < dist2 || (d2 == dist2 && cells[i] < closest)) {
          dist2 = d2;
          closest = cells[i];
        }
      }
    }
  };

  // Fast case query for searching cell that contains points
  template<typename Coord_T, size_t Dim>
  struct fast_query_functor : public thrust::unary_function<typename ntuple<Coord_T, Dim>::tuple, int>
  {
    // Convenience typedefs
    typedef typename ntuple<Coord_T, Dim>::tuple Coord_Tuple;
    tuple_unary_apply<Coord_Tuple, Coord_Tuple, shift_coord<Coord_T>, Dim> create_coord;

    // Grid functors
    clip_cell_functor<Dim> clip_cell;
    cell_containing_point_functor<Coord_T, Dim> cell_containing_point;
    query_cell_functor<Coord_T, Dim> query_cell;

    // Default constructor
    fast_query_functor() : clip_cell(clip_cell_functor<Dim>()),
                           cell_containing_point(cell_containing_point_functor<Coord_T, Dim>()),
                           query_cell(query_cell_functor<Coord_T, Dim>()) {}

    fast_query_functor(clip_cell_functor<Dim> clip_cell,
                       cell_containing_point_functor<Coord_T, Dim> cell_containing_point,
                       query_cell_functor<Coord_T, Dim> query_cell)
      : clip_cell(clip_cell), cell_containing_point(cell_containing_point), query_cell(query_cell) {}
    
    __host__ __device__
    int operator()(const Coord_Tuple& q) {
      int queryint = query_cell.point_to_id(q);
      int closestpt = -1;
      double dist2 = -1;
      query_cell(queryint, q, closestpt, dist2);
      // Calculate distance with extra for roundoff errors
      const double distf = sqrt(dist2) * 1.00001;

      // Find cells within distf of queried cell
      shift_coord<Coord_T> near_coord_lo(distf, true);
      shift_coord<Coord_T> near_coord_hi(distf, false);
      
      Coord_Tuple lopt = create_coord(q, near_coord_lo);
      Coord_Tuple hipt = create_coord(q, near_coord_hi);

      // Find endpoint cells that need to be searched
      Cell<Dim> locell(cell_containing_point(lopt));
      Cell<Dim> hicell(cell_containing_point(hipt));

      clip_cell(locell);
      clip_cell(hicell);

      Cell<Dim> qcell(cell_containing_point(q));
      // If cells equal query cell, already done
      if (locell == qcell && hicell == qcell) {
        return closestpt;
      }
      
      int close2 = -1;
      double d2 = -1;
      
      short int coords[Dim];
      size_t index = Dim-1;

      // While loop causes major slowdown, precompute number of iterations
      int itrs = 1;
      for (size_t i=0; i<Dim; ++i) {
        coords[i] = locell[i];
        itrs *= (hicell[i] - locell[i] + 1);
      }
      // Iterative nested for loop to query nearby cells
      for (int i=0; i<itrs; ++i) {
        queryint = query_cell.point_to_id.cell_to_id(Cell<Dim>(coords));
        query_cell(queryint, q, close2, d2);
        if (close2 != -1 && (d2 < dist2 || (d2 == dist2 && close2 < closestpt))) {
          closestpt = close2;
          dist2 = d2;
        }
        coords[Dim-1]++;

        while (coords[index] > hicell[index]) {
          if (index != 0) {
            coords[index] = locell[index];
            index--;
            coords[index]++;
          }
          else {
            break;
          }
        }
        index = Dim-1;
      }
      return closestpt;
    }
  };

  // Slow case query for searching cells around query point
  template<typename Coord_T, size_t Dim>
  struct slow_query_functor : public thrust::unary_function<typename ntuple<Coord_T, Dim>::tuple, int>
  {
    // Convenience typedefs
    typedef typename ntuple<Coord_T, Dim>::tuple Coord_Tuple;
    typedef typename Cell<Dim>::Cell_Index_T Cell_Index_T;
    typedef typename ntuple<Cell_Index_T, Dim>::tuple Cell_Tuple;
    typedef thrust::device_ptr<Cell_Index_T> Cell_Ptr;
    typedef typename ntuple<Cell_Ptr, Dim>::tuple Cell_Ptr_Tuple;
    typedef typename ntuple<int, Dim>::tuple Int_Tuple;
    ntuple<int, Dim> Int_Ntuple;
    ntuple<Cell_Ptr, Dim> Cell_Ptr_Ntuple;
    
    int ncellsearch;
    Cell_Ptr_Tuple cellsearch;
    thrust::device_ptr<int> cellstop;
    sign<Dim> signs;
    perm<Dim> perms;
    Cell_Ptr_Tuple pts;

    // Tuple operation functors
    get_point<Cell_Ptr, Cell_Index_T> point;
    tuple_binary_apply<Cell_Ptr_Tuple, Int_Tuple,
      Cell_Tuple, get_point<Cell_Ptr, Cell_Index_T>, Dim> pget;

    // Grid functors
    check_cell_functor<Dim> check_cell;
    cell_containing_point_functor<Coord_T, Dim> cell_containing_point;
    query_cell_functor<Coord_T, Dim> query_cell;

    // Default constructor
    slow_query_functor() : ncellsearch(0),
                           cellstop(thrust::device_ptr<int>()),
                           check_cell(check_cell_functor<Dim>()),
                           cell_containing_point(cell_containing_point_functor<Coord_T, Dim>()),
                           query_cell(query_cell_functor<Coord_T, Dim>()) {
      signs = sign<Dim>();
      perms = perm<Dim>();
      Cell_Ptr ptrs[Dim];
      for (size_t i=0; i<Dim; ++i) {ptrs[i] = Cell_Ptr();}
      pts = Cell_Ptr_Ntuple.make_tuple(ptrs);
    }

    // Constructor
    slow_query_functor(int ncellsearch,
                       Cell_Ptr_Tuple cellsearch,
                       thrust::device_ptr<int> cellstop,
                       check_cell_functor<Dim> check_cell,
                       cell_containing_point_functor<Coord_T, Dim> cell_containing_point,
                       query_cell_functor<Coord_T, Dim> query_cell)
      : ncellsearch(ncellsearch), cellsearch(cellsearch), cellstop(cellstop), check_cell(check_cell),
        cell_containing_point(cell_containing_point), query_cell(query_cell) {
      signs = sign<Dim>();
      perms = perm<Dim>();
    }

    // Get tuple of cell at point i
    __host__ __device__
    Cell_Tuple point_at(int i) {
      int id[Dim];
      for (int d=0; d<Dim; ++d) {id[d] = i;}
      Int_Tuple id_tup = Int_Ntuple.make_tuple(id);
      return pget(cellsearch, id_tup, point);
    }

    __host__ __device__
    int operator()(const Coord_Tuple& q) {
      Cell<Dim> qcell = cell_containing_point(q);
      int queryint = query_cell.point_to_id.cell_to_id(qcell);
      int closestpt = -1;
      double dist2 = -1;

      int nstop(ncellsearch);
      bool found(false);

      // Spiral out from starting cell
      for (int isort=0; isort<nstop; ++isort) {
        int close2;
        double d2;
        const Cell<Dim> s(point_at(isort));

        // Permute possible signs
        for (int isign = 0; isign < 1<<Dim; ++isign) {
          // Skip duplicates for zeros
          bool skip(false);
          for (size_t i=0; i<Dim; ++i) {
            if (s[i]==0 && signs[isign][i]==-1) {
              skip = true;
              break;
            }
          }
          if (skip) continue;
          
          const Cell<Dim> s2(s*signs[isign]);

          // Loop through permutations of indices
          for (int iperm=0; iperm<factorial<Dim>::value; ++iperm) {
            // Could skip duplicate permutations, not implemented
            
            Cell<Dim> s3;
            for (size_t i=0; i<Dim; ++i) {
              s3[i] = s2[perms[iperm][i]];
            }
            
            const Cell<Dim> c2(qcell+s3);
            if (!check_cell(c2)) continue;
            int cell_id(query_cell.point_to_id.cell_to_id(c2));
            query_cell(cell_id, q, close2, d2);
            if (close2 < 0) continue;

            if (dist2 == -1 || d2 < dist2 || (d2 == dist2 && close2 < closestpt)) {
              dist2 = d2;
              closestpt = close2;
              if (!found) {
                found = true;
                nstop = cellstop[isort];
                if (nstop >= ncellsearch) {
                  // Exit out of loops
                  return closestpt;
                }
              }
            }
          }
        }
      }
      return closestpt;
    }
  };

  template<typename Coord_T, size_t Dim>
  class Grid_T {
  public:
    // Typedefs from Point_Vector class
    typedef typename Point_Vector<Coord_T, Dim>::Coord_Tuple Coord_Tuple;
    typedef typename Point_Vector<Coord_T, Dim>::Coord_Tuple_Iterator Coord_Tuple_Iterator;
    typedef typename ntuple<double, Dim>::tuple Double_Tuple;

    int ng;
    int ngd;
    double r_cell;
    Double_Tuple d_cell;
    int nfixpts;
    Point_Vector<Coord_T, Dim>* pts;
    thrust::device_vector<int> cells;
    thrust::device_vector<int> base;
    cellsearchorder<Dim> cellsearch;

    #ifdef STATS
    thrust::device_vector<int> Num_Points_Per_Cell;
    int Min_Points_Per_Cell;
    int Max_Points_Per_Cell;
    float Avg_Points_Per_Cell;
    int Num_Fast_Queries;
    int Num_Slow_Queries;
    int Num_Exhaustive_Queries;
    static const int Max_Cells_Searched = 1000;
    vector<int> Num_Cells_Searched;
    int Total_Cells_Searched;
    static const int Max_Points_Checked = 10000;
    vector<int> Num_Points_Checked;
    int Total_Points_Checked;
    int Points_Checked;
    #endif

    // Functors
    check_cell_functor<Dim> check_cell;
    clip_cell_functor<Dim> clip_cell;
    cell_containing_point_functor<Coord_T, Dim> cell_containing_point;
    cell_to_id_functor<Dim> cell_to_id;
    point_to_id_functor<Coord_T, Dim> point_to_id;
    num_points_in_cell_id_functor num_points_in_cell_id;
    query_cell_functor<Coord_T, Dim> query_cell;
    fast_query_functor<Coord_T, Dim> fast_query;
    slow_query_functor<Coord_T, Dim> slow_query;

    int exhaustive_query(const Coord_Tuple& q) {
      typedef thrust::transform_iterator<distance2_functor<Coord_T, Dim>, Coord_Tuple_Iterator> dist2_itr;
      distance2_functor<Coord_T, Dim> distance2(q);
      dist2_itr begin(pts->begin(), distance2);
      dist2_itr end(pts->end(), distance2);
      dist2_itr result = thrust::min_element(begin, end);
      int closestpt = result - begin;
      return closestpt;
    }
  };
};
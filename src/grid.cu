#include "hip/hip_runtime.h"
#pragma once

#include <thrust/device_vector.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/extrema.h>

#include "point_vector.cu"
#include "functors.cu"
#include "tuple_utility.cu"

using namespace std;

namespace nearpt3 {

  // Check if this is a legal cell.
  template<size_t Dim>
  struct check_cell_functor
  {
    int ng;

    check_cell_functor() : ng(-1) {}
    check_cell_functor(int ng) : ng(ng) {}

    __host__ __device__
    bool operator()(const Cell<Dim>& a) const {
      for (int i=0; i<Dim; ++i) {
        if (a[i] < 0 || a[i] >= ng) return false;
      }
      return true;
    }
  };
  
  // clip is needed because roundoff errors may cause a number to be slightly outside the legal range
  template<size_t Dim>
  struct clip_cell_functor
  {
    int ng;

    clip_cell_functor() : ng(-1) {}
    clip_cell_functor(int ng) : ng(ng) {}

    __host__ __device__
    void operator()(Cell<Dim>& a) {
      for (size_t i=0; i<Dim; ++i) {
        if (a[i] < 0) a[i] = 0;
        if (a[i] >= ng) a[i] = ng-1;
      }
    }
  };

  template<typename Coord_T, size_t Dim>
  struct cell_containing_point_functor
    : public thrust::unary_function<typename ntuple<Coord_T, Dim>::tuple, Cell<Dim> >
  {
    typedef typename Cell<Dim>::Cell_Index_T Cell_Index_T;
    typedef typename ntuple<double, Dim>::tuple Double_Tuple;
    typedef typename ntuple<Coord_T, Dim>::tuple Coord_Tuple;
    typedef typename ntuple<Cell_Index_T, Dim>::tuple Cell_Tuple;
    double r_cell;
    Double_Tuple d_cell;
    // Only used for making tuple
    ntuple<double, Dim> Double_Ntuple;
    coord_to_cell_index<Coord_T, Cell_Index_T> cts;
    tuple_binary_apply<Coord_Tuple, Double_Tuple, Cell_Tuple,
      coord_to_cell_index<Coord_T, Cell_Index_T>, Dim> make_cell;

    cell_containing_point_functor() : r_cell(-1) {
      double d[Dim];
      for (int i=0; i<Dim; ++i) {d[i] = -1;}
      d_cell = Double_Ntuple.make_tuple(d);
    }

    cell_containing_point_functor(double r_cell, Double_Tuple d_cell)
      : r_cell(r_cell), d_cell(d_cell) {
      cts = coord_to_cell_index<Coord_T, Cell_Index_T>(r_cell);
    }

    __host__ __device__
    Cell<Dim> operator()(const Coord_Tuple& a) const {
      // short int i[Dim];
      // i[0] = static_cast<short int>(static_cast<double>(thrust::get<0>(a))*r_cell+thrust::get<0>(d_cell));
      // i[1] = static_cast<short int>(static_cast<double>(thrust::get<1>(a))*r_cell+thrust::get<1>(d_cell));
      // i[2] = static_cast<short int>(static_cast<double>(thrust::get<2>(a))*r_cell+thrust::get<2>(d_cell));
      // Cell<Dim> c(i);
      Cell<Dim> c(make_cell(a, d_cell, cts));
      return c;
    }
  };

  template<size_t Dim>
  struct cell_to_id_functor : public thrust::unary_function<Cell<Dim>, int>
  {
    int ng;

    cell_to_id_functor() : ng(-1) {}
    cell_to_id_functor(int ng) : ng(ng) {}
  
    __host__ __device__
    int operator()(const Cell<Dim>& c) const {
      int id = 0;
      for (int i=0; i<Dim; ++i) {
        if (c[i] < 0 || c[i] >= ng) return -1;
        id = static_cast<int>(c[i]) + ng * id;
      }
      return id;
    }
  };
  
  template<typename Coord_T, size_t Dim>
  struct point_to_id_functor : public thrust::unary_function<typename ntuple<Coord_T, Dim>::tuple, int>
  {
    typedef typename ntuple<Coord_T, Dim>::tuple Coord_Tuple;
    
    cell_containing_point_functor<Coord_T, Dim> cell_containing_point;
    cell_to_id_functor<Dim> cell_to_id;

    point_to_id_functor() : cell_containing_point(cell_containing_point_functor<Coord_T, Dim>()),
                            cell_to_id(cell_to_id_functor<Dim>()) {}
    
    point_to_id_functor(cell_containing_point_functor<Coord_T, Dim> cell_containing_point,
                        cell_to_id_functor<Dim> cell_to_id)
      : cell_containing_point(cell_containing_point), cell_to_id(cell_to_id) {}

    __host__ __device__
    int operator()(const Coord_Tuple& a) const {
      return cell_to_id(cell_containing_point(a));
    }
  };

  struct num_points_in_cell_id_functor : public thrust::unary_function<int, int>
  {
    thrust::device_ptr<int> base;

    num_points_in_cell_id_functor() : base(thrust::device_ptr<int>()) {}
    num_points_in_cell_id_functor(thrust::device_ptr<int> base) : base(base) {}

    __host__ __device__
    int operator()(const int& id) const {
      if (id < 0) return 0;
      return base[id+1] - base[id];
    }
  };


  template<typename Coord_T, size_t Dim>
  struct query_cell_functor
  {
    typedef typename ntuple<Coord_T, Dim>::tuple Coord_Tuple;
    typedef thrust::device_ptr<Coord_T> Coord_Ptr;
    typedef typename ntuple<Coord_Ptr, Dim>::tuple Coord_Ptr_Tuple;
    typedef typename ntuple<int, Dim>::tuple Int_Tuple;
    ntuple<Coord_Ptr, Dim> Coord_Ptr_Ntuple;
    ntuple<int, Dim> Int_Ntuple;
    get_point<Coord_Ptr, Coord_T> point;
    tuple_binary_apply<Coord_Ptr_Tuple, Int_Tuple,
      Coord_Tuple, get_point<Coord_Ptr, Coord_T>, Dim> pget;
    
    num_points_in_cell_id_functor num_points_in_cell_id;
    point_to_id_functor<Coord_T, Dim> point_to_id;
    thrust::device_ptr<int> base;
    thrust::device_ptr<int> cells;
    Coord_Ptr_Tuple pts;

    query_cell_functor() : num_points_in_cell_id(num_points_in_cell_id_functor()),
                           point_to_id(point_to_id_functor<Coord_T, Dim>()),
                           base(thrust::device_ptr<int>()),
                           cells(thrust::device_ptr<int>()) {
      Coord_Ptr ptrs[Dim];
      for (size_t i=0; i<Dim; ++i) {ptrs[i] = Coord_Ptr();}
      pts = Coord_Ptr_Ntuple.make_tuple(ptrs);
    }
    
    query_cell_functor(num_points_in_cell_id_functor num_points_in_cell_id,
                       point_to_id_functor<Coord_T, Dim> point_to_id,
                       thrust::device_ptr<int> base,
                       thrust::device_ptr<int> cells,
                       Coord_Ptr_Tuple pts)
      : num_points_in_cell_id(num_points_in_cell_id), point_to_id(point_to_id),
        base(base), cells(cells), pts(pts) {}

    __host__ __device__
    Coord_Tuple point_at(int i) {
      int id[Dim];
      for (int d=0; d<Dim; ++d) {id[d] = i;}
      Int_Tuple id_tup = Int_Ntuple.make_tuple(id);
      return pget(pts, id_tup, point);
    }
    
    __host__ __device__
    void operator()(const int &cell_id, const Coord_Tuple& q,
                    int &closest, double &dist2) {
      const int num_points(num_points_in_cell_id(cell_id));
      if (num_points <= 0) {
        closest = -1;
        dist2 = -1;
        return;
      }
      
      const int queryint(point_to_id(q));
      distance2_functor<Coord_T, Dim> distance2(q);
      int i = base[cell_id];
      closest = cells[i];
      dist2 = distance2(point_at(closest));
      while (i < base[cell_id+1]) {
        const double d2 = distance2(point_at(cells[i]));
        if (d2 < dist2 || (d2 == dist2 && cells[i] < closest)) {
          dist2 = d2;
          closest = cells[i];
        }
        ++i;
      }
    }
  };

  template<typename Coord_T, size_t Dim>
  struct fast_query_functor : public thrust::unary_function<typename ntuple<Coord_T, Dim>::tuple, int>
  {
    typedef typename ntuple<Coord_T, Dim>::tuple Coord_Tuple;
    tuple_unary_apply<Coord_Tuple, Coord_Tuple, near_cell<Coord_T>, Dim> dc;
    
    clip_cell_functor<Dim> clip_cell;
    cell_containing_point_functor<Coord_T, Dim> cell_containing_point;
    query_cell_functor<Coord_T, Dim> query_cell;

    fast_query_functor() : clip_cell(clip_cell_functor<Dim>()),
                           cell_containing_point(cell_containing_point_functor<Coord_T, Dim>()),
                           query_cell(query_cell_functor<Coord_T, Dim>()) {}

    fast_query_functor(clip_cell_functor<Dim> clip_cell,
                       cell_containing_point_functor<Coord_T, Dim> cell_containing_point,
                       query_cell_functor<Coord_T, Dim> query_cell)
      : clip_cell(clip_cell), cell_containing_point(cell_containing_point), query_cell(query_cell) {}
    
    __host__ __device__
    int operator()(const Coord_Tuple& q) {
      int queryint = query_cell.point_to_id(q);
      int closestpt = -1;
      double dist2 = -1;
      query_cell(queryint, q, closestpt, dist2);
      const double distf = sqrt(dist2) * 1.00001;
      // Coord_Tuple lopt(thrust::make_tuple(clamp_USI(static_cast<double>(thrust::get<0>(q)) - distf),
      //                                     clamp_USI(static_cast<double>(thrust::get<1>(q)) - distf),
      //                                     clamp_USI(static_cast<double>(thrust::get<2>(q)) - distf)));
      // Coord_Tuple hipt(thrust::make_tuple(clamp_USI(static_cast<double>(thrust::get<0>(q)) + distf + 1.0),
      //                                     clamp_USI(static_cast<double>(thrust::get<1>(q)) + distf + 1.0),
      //                                     clamp_USI(static_cast<double>(thrust::get<2>(q)) + distf + 1.0)));
      near_cell<Coord_T> near_cell_lo(distf, true);
      near_cell<Coord_T> near_cell_hi(distf, false);
      
      Coord_Tuple lopt = dc(q, near_cell_lo);
      Coord_Tuple hipt = dc(q, near_cell_hi);
      
      Cell<Dim> locell(cell_containing_point(lopt));
      Cell<Dim> hicell(cell_containing_point(hipt));

      clip_cell(locell);
      clip_cell(hicell);

      Cell<Dim> qcell(cell_containing_point(q));
      if (locell == qcell && hicell == qcell) {
        return closestpt;
      }
      int close2 = -1;
      double d2 = -1;
      // for (Coord_T x=locell[0]; x<=hicell[0]; ++x) {
      //   for (Coord_T y=locell[1]; y<=hicell[1]; ++y) {
      //     for (Coord_T z=locell[2]; z<=hicell[2]; ++z) {
      //       queryint = query_cell.point_to_id.cell_to_id(Cell<Dim>(x, y, z));
      //       query_cell(queryint, q, close2, d2);
      //       if (close2 != -1 && (d2 < dist2 || (d2 == dist2 && close2 < closestpt))) {
      //         closestpt = close2;
      //         dist2 = d2;
      //       }
      //     }
      //   }
      // }
      
      // Nested loop traversal, modified from this code:
      // http://stackoverflow.com/questions/18732974/c-dynamic-number-of-nested-for-loops-without-recursion
      short int coords[Dim];
      size_t index = 0;

      // While loop causes major slowdown, precompute number of iterations
      int itrs = 1;
      for (size_t i=0; i<Dim; ++i) {
        coords[i] = locell[i];
        itrs *= (hicell[i] - locell[i] + 1);
      }
      for (int i=0; i<itrs; ++i) {
        queryint = query_cell.point_to_id.cell_to_id(Cell<Dim>(coords));
        query_cell(queryint, q, close2, d2);
        if (close2 != -1 && (d2 < dist2 || (d2 == dist2 && close2 < closestpt))) {
          closestpt = close2;
          dist2 = d2;
        }
        coords[0]++;

        while (coords[index] == hicell[index]) {
          if (index != Dim - 1) {
            coords[index] = locell[index];
            index++;
            coords[index]++;
          }
          else {
            break;
          }
        }
        index = 0;
      }

      return closestpt;
    }
  };

  template<typename Coord_T, size_t Dim>
  struct slow_query_functor : public thrust::unary_function<typename ntuple<Coord_T, Dim>::tuple, int>
  {
    typedef typename ntuple<Coord_T, Dim>::tuple Coord_Tuple;
    typedef typename Cell<Dim>::Cell_Index_T Cell_Index_T;
    typedef typename ntuple<Cell_Index_T, Dim>::tuple Cell_Tuple;
    typedef thrust::device_ptr<Cell_Index_T> Cell_Ptr;
    typedef typename ntuple<Cell_Ptr, Dim>::tuple Cell_Ptr_Tuple;
    typedef typename ntuple<int, Dim>::tuple Int_Tuple;
    ntuple<int, Dim> Int_Ntuple;
    ntuple<Cell_Ptr, Dim> Cell_Ptr_Ntuple;
    
    int ncellsearch;
    Cell_Ptr_Tuple cellsearch;
    thrust::device_ptr<int> cellstop;
    sign<Dim> signs;
    perm<Dim> perms;
    Cell_Ptr_Tuple pts;
    get_point<Cell_Ptr, Cell_Index_T> point;
    tuple_binary_apply<Cell_Ptr_Tuple, Int_Tuple,
      Cell_Tuple, get_point<Cell_Ptr, Cell_Index_T>, Dim> pget;

    check_cell_functor<Dim> check_cell;
    cell_containing_point_functor<Coord_T, Dim> cell_containing_point;
    query_cell_functor<Coord_T, Dim> query_cell;

    slow_query_functor() : ncellsearch(0),
                           cellstop(thrust::device_ptr<int>()),
                           check_cell(check_cell_functor<Dim>()),
                           cell_containing_point(cell_containing_point_functor<Coord_T, Dim>()),
                           query_cell(query_cell_functor<Coord_T, Dim>()) {
      signs = sign<Dim>();
      perms = perm<Dim>();
      Cell_Ptr ptrs[Dim];
      for (size_t i=0; i<Dim; ++i) {ptrs[i] = Cell_Ptr();}
      pts = Cell_Ptr_Ntuple.make_tuple(ptrs);
    }

    slow_query_functor(int ncellsearch,
                       Cell_Ptr_Tuple cellsearch,
                       thrust::device_ptr<int> cellstop,
                       check_cell_functor<Dim> check_cell,
                       cell_containing_point_functor<Coord_T, Dim> cell_containing_point,
                       query_cell_functor<Coord_T, Dim> query_cell)
      : ncellsearch(ncellsearch), cellsearch(cellsearch), cellstop(cellstop), check_cell(check_cell),
        cell_containing_point(cell_containing_point), query_cell(query_cell) {
      signs = sign<Dim>();
      perms = perm<Dim>();
    }

    __host__ __device__
    Cell_Tuple point_at(int i) {
      int id[Dim];
      for (int d=0; d<Dim; ++d) {id[d] = i;}
      Int_Tuple id_tup = Int_Ntuple.make_tuple(id);
      return pget(cellsearch, id_tup, point);
    }

    __host__ __device__
    int operator()(const Coord_Tuple& q) {
      // const int sign3[8][3] = {{1,1,1},{1,1,-1},{1,-1,1},{1,-1,-1},
      //                          {-1,1,1},{-1,1,-1},{-1,-1,1},{-1,-1,-1}};
      // const int perm3[6][3] = {{0,1,2},{0,2,1},{1,0,2},{1,2,0},{2,0,1},{2,1,0}};

      Cell<Dim> qcell = cell_containing_point(q);
      int queryint = query_cell.point_to_id.cell_to_id(qcell);
      int closestpt = -1;
      double dist2 = -1;

      int nstop(ncellsearch);
      bool found(false);

      for (int isort=0; isort<nstop; ++isort) {
        int close2;
        double d2;
        const Cell<Dim> s(point_at(isort));

        for (int isign=0; isign<1<<Dim; ++isign) {
          bool skip(false);
          for (size_t i=0; i<Dim; ++i) {
            if (s[i]==0 && signs[isign][i]==-1) {
              skip = true;
              break;
            }
          }
          if (skip) continue;
          
          const Cell<Dim> s2(s*signs[isign]);

          for (int iperm=0; iperm<factorial<Dim>::value; ++iperm) {
            // switch (iperm) {
            // case 1:
            //   if (s[1]==s[2]) continue;
            //   break;
            // case 2: 
            //   if (s[0]==s[1]) continue;
            //   break;
            // case 3:
            // case 4:
            //   if (s[0]==s[1] && s[0]==s[2]) continue;
            //   break;
            // case 5:
            //   if (s[0]==s[2]) continue;
            //   break;
            // }
            if (iperm > 0 && (s[iperm % Dim] == s[(iperm-1) % Dim])) continue;
            
            Cell<Dim> s3;
            //(s2[perms[iperm][0]], s2[perms[iperm][1]], s2[perms[iperm][2]]);
            for (size_t i=0; i<Dim; ++i) {
              s3[i] = s2[perms[iperm][i]];
            }
            
            const Cell<Dim> c2(qcell+s3);
            if (!check_cell(c2)) continue;
            int cell_id(query_cell.point_to_id.cell_to_id(c2));
            query_cell(cell_id, q, close2, d2);
            if (close2 < 0) continue;

            if (dist2 == -1 || d2 < dist2 || (d2 == dist2 && close2 < closestpt)) {
              dist2 = d2;
              closestpt = close2;
              if (!found) {
                found = true;
                nstop = cellstop[isort];
                if (nstop >= ncellsearch) {
                  iperm = 1<<Dim;
                  isign = factorial<Dim>::value;
                  isort = nstop;
                }
              }
            }
          }
        }
      }
      return closestpt;
    }
  };

  template<typename Coord_T, size_t Dim>
  class Grid_T {
  public:
    // Typedefs from Point_Vector class
    typedef typename Point_Vector<Coord_T, Dim>::Coord_Tuple Coord_Tuple;
    typedef typename Point_Vector<Coord_T, Dim>::Coord_Tuple_Iterator Coord_Tuple_Iterator;
    typedef typename ntuple<double, Dim>::tuple Double_Tuple;

    int ng;
    int ng3;
    double r_cell;
    Double_Tuple d_cell;
    int nfixpts;
    Point_Vector<Coord_T, Dim>* pts;
    thrust::device_vector<int> cells;
    thrust::device_vector<int> base;
    //thrust::device_vector<int> cellsearch;
    cellsearchcreate<Dim> cellsearch;

    #ifdef STATS
    thrust::device_vector<int> Num_Points_Per_Cell;
    int Min_Points_Per_Cell;
    int Max_Points_Per_Cell;
    float Avg_Points_Per_Cell;
    int Num_Fast_Queries;
    int Num_Slow_Queries;
    int Num_Exhaustive_Queries;
    static const int Max_Cells_Searched = 1000;
    vector<int> Num_Cells_Searched;
    int Total_Cells_Searched;
    static const int Max_Points_Checked = 10000;
    vector<int> Num_Points_Checked;
    int Total_Points_Checked;
    int Points_Checked;
    #endif

    // Functors
    check_cell_functor<Dim> check_cell;
    clip_cell_functor<Dim> clip_cell;
    cell_containing_point_functor<Coord_T, Dim> cell_containing_point;
    cell_to_id_functor<Dim> cell_to_id;
    point_to_id_functor<Coord_T, Dim> point_to_id;
    num_points_in_cell_id_functor num_points_in_cell_id;
    query_cell_functor<Coord_T, Dim> query_cell;
    fast_query_functor<Coord_T, Dim> fast_query;
    slow_query_functor<Coord_T, Dim> slow_query;

    int exhaustive_query(const Coord_Tuple& q) {
      typedef thrust::transform_iterator<distance2_functor<Coord_T, Dim>, Coord_Tuple_Iterator> dist2_itr;
      distance2_functor<Coord_T, Dim> distance2(q);
      dist2_itr begin(pts->begin(), distance2);
      dist2_itr end(pts->end(), distance2);
      dist2_itr result = thrust::min_element(begin, end);
      int closestpt = result - begin;
      return closestpt;
    }
  };
};
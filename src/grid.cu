#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>

#include <boost/multi_array.hpp>
#include <iostream>

#include "points.cu"
#include "functors.cu"
#include "cell.cpp"

using namespace std;
using boost::array;

// Print an expression's name then its value, possibly followed by a comma or endl.  
// Ex: cout << PRINTC(x) << PRINTN(y);

#define PRINT(arg)  #arg "=" << (arg)
#define PRINTC(arg)  #arg "=" << (arg) << ", "
#define PRINTN(arg)  #arg "=" << (arg) << endl

// clamp_USI     Convert to an unsigned short int while clamping

template <typename T>
unsigned short int clamp_USI(T a) {
  const unsigned short int m(numeric_limits<unsigned short int>::max());
  const T mm(static_cast<T>(m));
  return  
    static_cast<unsigned short int>(a > mm ? mm : (a > 0 ? static_cast<unsigned short int>(a) : 0));
}

template <typename T>
void write(ostream &o, const thrust::tuple<T, T, T>& c) {
  o << "(" << thrust::get<0>(c) << "," << thrust::get<1>(c) << "," << thrust::get<2>(c) << ")";
}

template <typename T>
void write(ostream &o, const array<T,3>& c) {
    o << "(" << c[0] << "," << c[1] << "," << c[2] << ")";
}

template<typename Coord_T> 
ostream &operator<<(ostream &o, const array<Coord_T,3> &c) {
  o << '(' << c[0] << ',' << c[1] << ',' << c[2] << ')';
  return o;
}

namespace nearpt3 {

  template<typename Coord_T>
  class Grid_T {
    typedef thrust::tuple<Coord_T, Coord_T, Coord_T> Coord3;

    typedef thrust::device_vector<Coord_T> Coord_Vector;
    typedef typename Coord_Vector::iterator Coord_Iterator;
    typedef thrust::tuple<Coord_Iterator, Coord_Iterator, Coord_Iterator> Coord_Iterator_Tuple;
    typedef thrust::zip_iterator<Coord_Iterator_Tuple> Coord_3_Iterator;

  public:
    int ng;
    int ng3;
    double r_cell;
    array<double,3> d_cell;
    int nfixpts;
    Points_T<Coord_T>* pts;
    thrust::device_vector<int> cells;
    thrust::device_vector<int> base;

    // Check if this is a legal cell.
    bool check(const Cell3 a) const {
      if (a[0] < 0 || a[0] >= ng ) return false;
      if (a[1] < 0 || a[1] >= ng ) return false;
      if (a[2] < 0 || a[2] >= ng ) return false;
      return true;
    }

    // clip is needed because roundoff errors may cause a number to be slightly outside the legal range. 
    void clip(Cell3 &a) {

      //    a[0] = min(ng-1, max(0, a[0]));

      if (a[0] < 0) a[0] = 0;
      if (a[0] >= ng) a[0] = ng-1;
      if (a[1] < 0) a[1] = 0;
      if (a[1] >= ng) a[1] = ng-1;
      if (a[2] < 0) a[2] = 0;
      if (a[2] >= ng) a[2] = ng-1;
    }
    
    // Compute_Cell_Containing_Point: Return the cell number containing point p.  Note the p is a
    // array<Coord_T, 3>, not an id of a point.  This is necessary since
    // Compute_Cell_Containing_Point is used for both fixed and query points.

    const Cell3 Compute_Cell_Containing_Point(const array<Coord_T,3> p) {
      const short int ix = static_cast<short int>(static_cast<double>(p[0])*r_cell+d_cell[0]);   // This must truncate not round.
      const short int iy = static_cast<short int>(static_cast<double>(p[1])*r_cell+d_cell[1]);
      const short int iz = static_cast<short int>(static_cast<double>(p[2])*r_cell+d_cell[2]);
      Cell3 c(ix, iy, iz);
      return c;
    }
    
    int point_to_id(const int& n) {
      Coord_3_Iterator p = pts->begin();
      int ix = static_cast<short int>(static_cast<double>(thrust::get<0>(p[n]))*r_cell+d_cell[0]);
      int iy = static_cast<short int>(static_cast<double>(thrust::get<1>(p[n]))*r_cell+d_cell[1]);
      int iz = static_cast<short int>(static_cast<double>(thrust::get<2>(p[n]))*r_cell+d_cell[2]);

      if (ix < 0 || ix >= ng || iy < 0 || iy >= ng || iz < 0 || iz >= ng) return -1;
      return (ix*ng + iy)*ng + iz;
    }

    int qpoint_to_id(const array<Coord_T, 3> q) {
      int ix = static_cast<short int>(static_cast<double>(q[0])*r_cell+d_cell[0]);
      int iy = static_cast<short int>(static_cast<double>(q[1])*r_cell+d_cell[1]);
      int iz = static_cast<short int>(static_cast<double>(q[2])*r_cell+d_cell[2]);

      if (ix < 0 || ix >= ng || iy < 0 || iy >= ng || iz < 0 || iz >= ng) return -1;
      return (ix*ng + iy)*ng + iz;
    }
    
    int cellid_to_int(const Cell3 a) const { 
      if (a[0]<0 || a[0] >=ng || a[1]<0 || a[1] >=ng || a[2]<0 || a[2] >=ng) return -1;
      return  (static_cast<int> (a[0])*ng + static_cast<int>(a[1]))*ng + a[2]; 
    }

    int num_points_id(const int id) {
      if (id<0) return 0;
      return base[id+1] - base[id];
    }

    void querythiscell(const Cell3 thiscell, const array<Coord_T, 3> q,
                       int &closestpt, double &dist2) {
      const int queryint(qpoint_to_id(q));
      const int npitc(num_points_id(queryint));
      if (npitc<=0) {
        closestpt = -1;
        dist2 = numeric_limits<double>::max();
        return;
      }
      typedef thrust::device_vector<int>::iterator IntItr;
      typedef thrust::permutation_iterator<Coord_3_Iterator, IntItr> PermItr;
      typedef thrust::transform_iterator<distance2_functor<Coord3>, PermItr> dist2_itr;
      PermItr ptsbegin(pts->begin(), cells.begin());
      dist2_itr begin(ptsbegin + base[queryint],
                      distance2_functor<Coord3>(q[0], q[1], q[2]));
      dist2_itr end(ptsbegin + base[queryint+1] - 1,
                    distance2_functor<Coord3>(q[0], q[1], q[2]));
      dist2_itr result = thrust::min_element(begin, end);
      closestpt = cells[result - begin + base[queryint]];
      dist2 = *result;
      return;
    }

    int Query_Fast_Case(const array<Coord_T, 3> q) {
      const int queryint(qpoint_to_id(q));
      const int npitc(num_points_id(queryint));

#ifdef DEBUG
      cout << PRINTC(q[0]) << PRINTC(q[1]) << PRINTN(q[2]);
      cout << PRINTC(queryint) << PRINTN(npitc);
      cout << PRINTC(base[queryint]) << PRINTN(base[queryint+1]);
#endif
      
      if (npitc<=0) return -1; // No points in this cell

      int closestpt = -1;

      // Thrust iterator black magic
      typedef thrust::device_vector<int>::iterator IntItr;
      typedef thrust::permutation_iterator<Coord_3_Iterator, IntItr> PermItr;
      typedef thrust::transform_iterator<distance2_functor<Coord3>, PermItr> dist2_itr;
      PermItr ptsbegin(pts->begin(), cells.begin());
      dist2_itr begin(ptsbegin + base[queryint],
                      distance2_functor<Coord3>(q[0], q[1], q[2]));
      dist2_itr end(ptsbegin + base[queryint+1] - 1,
                    distance2_functor<Coord3>(q[0], q[1], q[2]));
      
      Coord_3_Iterator i(pts->begin());
      i += cells[base[queryint]];
      Coord_T x = thrust::get<0>(*i);
      Coord_T y = thrust::get<1>(*i);
      Coord_T z = thrust::get<2>(*i);
#ifdef DEBUG
      cout << PRINTC(cells[base[queryint]]) << PRINTN(cells[base[queryint+1]-1]);
      cout << PRINTC(*begin) << PRINTN(*end);
      cout << PRINTC(x) << PRINTC(y) << PRINTN(z);
#endif
      
      dist2_itr result = thrust::min_element(begin, end);
      double dist2 = *result;

      closestpt = cells[result - begin + base[queryint]];
      //closestpt = cells[result - begin];
      
      const double distf = sqrt(dist2) * 1.00001;

#ifdef DEBUG
      cout << PRINTC(closestpt) << PRINTC(end - begin) <<  PRINTC(result - begin) <<
        PRINTC(result - begin + base[queryint]) << PRINTC(*result) << PRINTN(distf);
#endif

      array<Coord_T, 3> lopt, hipt;
      for (int i=0; i<3; ++i) {
        lopt[i] = static_cast<unsigned short int> (clamp_USI(static_cast<double>(q[i]) - distf));
        hipt[i] = static_cast<unsigned short int> (clamp_USI(static_cast<double>(q[i]) + distf + 1.0));
      }

      Cell3 locell(Compute_Cell_Containing_Point(lopt));
      Cell3 hicell(Compute_Cell_Containing_Point(hipt));

      clip(locell);
      clip(hicell);

      cout << PRINTC(lopt) << PRINTN(hipt);
      Cell3 qcell(Compute_Cell_Containing_Point(q));
      if (locell == qcell && hicell == qcell) return closestpt;

      
      for (Coord_T x=locell[0]; x<=hicell[0]; x++) {
        for (Coord_T y=locell[1]; y<=hicell[1]; y++) {
          // Do a whole z-row of cells at once.
          const int i01 = (static_cast<int>(x)*ng + static_cast<int>(y))*ng;
          const int i0 = i01 + locell[2];
          const int i1 = i01 + hicell[2];
#ifdef DEBUG
          cout << PRINTC(x) << PRINTC(y) << PRINTC(i01) <<
            PRINTC(i0) << PRINTC(i1) << PRINTC(base[i0]) << PRINTN(base[i1+1]);
#endif

          dist2_itr b(ptsbegin + base[i0],
                      distance2_functor<Coord3>(q[0], q[1], q[2]));
          cout << "A" << endl;
          dist2_itr e(ptsbegin + base[i1+1] - 1,
                      distance2_functor<Coord3>(q[0], q[1], q[2]));
          cout << "B" << endl;
          cout << PRINTC(*b) << PRINTN(*e);
          dist2_itr r = thrust::min_element(b, e);
          double d2 = *r;
          cout << PRINTC(base[i1+1]-1) << PRINTC(d2) << PRINTC(r - b) << PRINTC(e - b) << PRINTN(r - b + base[i0]);
          if (d2 < dist2 || (d2==dist2 && cells[r - b + base[i0]]<closestpt)) {
            cout << "C" << endl;
            dist2 = d2;
            closestpt = cells[r - b + base[i0]];
            cout << PRINTC(d2) << PRINTC(r - b + base[i0]) << PRINTN(cells[r - b + base[i0]]);
            cout << "D" << endl;
          }
        }
      }

      cout << PRINTC(closestpt) << PRINTN(dist2);
      cout << endl;
    
      return closestpt;
    }
  };

};
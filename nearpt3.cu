#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/copy.h>

#include <algorithm>
#include <boost/multi_array.hpp> 
#include <iomanip>
#include <iostream>
#include <fstream>
#include <math.h>
#include <sys/resource.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <sys/times.h>
#include <sys/types.h>
#include <time.h>
#include <unistd.h>

using namespace std;
using boost::array;

#define DEBUG

// Print an expression's name then its value, possibly followed by a comma or endl.  
// Ex: cout << PRINTC(x) << PRINTN(y);

#define PRINT(arg)  #arg "=" << (arg)
#define PRINTC(arg)  #arg "=" << (arg) << ", "
#define PRINTN(arg)  #arg "=" << (arg) << endl

// clamp_USI     Convert to an unsigned short int while clamping

template <typename T>
unsigned short int clamp_USI(T a) {
  const unsigned short int m(numeric_limits<unsigned short int>::max());
  const T mm(static_cast<T>(m));
  return  
    static_cast<unsigned short int>(a > mm ? mm : (a > 0 ? static_cast<unsigned short int>(a) : 0));
}

// From thrust example code
template <typename Iterator>
class strided_range
{
    public:

    typedef typename thrust::iterator_difference<Iterator>::type difference_type;

    struct stride_functor : public thrust::unary_function<difference_type,difference_type>
    {
        difference_type stride;

        stride_functor(difference_type stride)
            : stride(stride) {}

        __host__ __device__
        difference_type operator()(const difference_type& i) const
        { 
            return stride * i;
        }
    };

    typedef typename thrust::counting_iterator<difference_type>                   CountingIterator;
    typedef typename thrust::transform_iterator<stride_functor, CountingIterator> TransformIterator;
    typedef typename thrust::permutation_iterator<Iterator,TransformIterator>     PermutationIterator;

    // type of the strided_range iterator
    typedef PermutationIterator iterator;

    // construct strided_range for the range [first,last)
    strided_range(Iterator first, Iterator last, difference_type stride)
        : first(first), last(last), stride(stride) {}
   
    iterator begin(void) const
    {
        return PermutationIterator(first, TransformIterator(CountingIterator(0), stride_functor(stride)));
    }

    iterator end(void) const
    {
        return begin() + ((last - first) + (stride - 1)) / stride;
    }
    
    protected:
    Iterator first;
    Iterator last;
    difference_type stride;
};

template<typename T>
struct point_to_id_functor : public thrust::unary_function<T, int>
{
  const int ng;
  const double r_cell;
  const double d1;
  const double d2;
  const double d3;

  point_to_id_functor(int ng, double r_cell, double d1, double d2, double d3)
    : ng(ng), r_cell(r_cell), d1(d1), d2(d2), d3(d3) { }
  
  __host__ __device__
  int operator()(const T& a) const {
    int ix = static_cast<short int>(static_cast<double>(thrust::get<0>(a))*r_cell+d1);
    int iy = static_cast<short int>(static_cast<double>(thrust::get<1>(a))*r_cell+d2);
    int iz = static_cast<short int>(static_cast<double>(thrust::get<2>(a))*r_cell+d3);

    if (ix < 0 || ix >= ng || iy < 0 || iy >= ng || iz < 0 || iz >= ng) return -1;
    return (ix*ng + iy)*ng + iz;
  }
};

template<typename T>
struct distance2_functor : public thrust::unary_function<T, double>
{
  const int x;
  const int y;
  const int z;

  distance2_functor(int x, int y, int z) : x(x), y(y), z(z) { }

  __host__ __device__
  double square(const double t) const {
    return t * t;
  }
  
  __host__ __device__
  double operator()(const T& a) const {
    return (square((thrust::get<0>(a)-x)) +
            square((thrust::get<1>(a)-y)) +
            square((thrust::get<2>(a)-z)));
  }
};

template <typename T>
void write(ostream &o, const thrust::tuple<T, T, T>& c) {
  o << "(" << thrust::get<0>(c) << "," << thrust::get<1>(c) << "," << thrust::get<2>(c) << ")";
}

template <typename T>
void write(ostream &o, const array<T,3>& c) {
    o << "(" << c[0] << "," << c[1] << "," << c[2] << ")";
}


namespace nearpt3 {
	double ng_factor = 1.6;

  // cellsearchorder:
  // First 3 elements of each row:  the order in which to search (one 48th-ant of the) cells adjacent to the current cell.
  // 4th element:   where, in cellsearchorder, to stop searching after the first point is found.
  const static int  cellsearchorder[][4] = {
#include "cellsearchorder"
  };
  // Number of cells in cellsearchorder (before expanding symmetries).
  const static int ncellsearchorder = 
    sizeof(nearpt3::cellsearchorder) / sizeof(nearpt3::cellsearchorder[0][0])/4;

  typedef short int Cell3_Index_T;
  class Cell3 {
  public:
    array<Cell3_Index_T,3> c;

    Cell3(const Cell3_Index_T x, const Cell3_Index_T y, const Cell3_Index_T z) {
      c[0] = x; c[1] = y; c[2] = z; }

    Cell3(const Cell3 &a) { c[0] = a[0]; c[1] = a[1]; c[2] = a[2]; }

    Cell3() { c[0] = -1; c[1] = -1; c[2] = -1; }

    Cell3_Index_T & operator[] (const int i)  {  return c[i];  }

    const Cell3_Index_T & operator[] (const int i) const {  return c[i];  }

    const Cell3 operator+(const Cell3 &d) const {
      Cell3 r;
      r[0] = c[0]+d[0];
      r[1] = c[1]+d[1];
      r[2] = c[2]+d[2];
      return r;
    }

    const Cell3 operator*(const int *d) const {
      Cell3 r;
      r[0] = c[0]*d[0];
      r[1] = c[1]*d[1];
      r[2] = c[2]*d[2];
      return r;
    }

    bool operator==(const Cell3 &d)  const {
      return c[0]==d[0] && c[1]==d[1] && c[2]==d[2];
    }

    //    const Cell3 operator*(const int *) const;
  };

  void write(ostream &o, const Cell3& c) {
    o << '(' << c[0] << ',' << c[1] << ',' << c[2] << ") ";
  }

  template<typename Coord_T>
  class Points_T {
    // Convenience Typedefs
    typedef thrust::device_vector<Coord_T> Coord_Vector;
    typedef typename Coord_Vector::iterator Coord_Iterator;
    typedef thrust::tuple<Coord_Iterator, Coord_Iterator, Coord_Iterator> Coord_Iterator_Tuple;
    typedef thrust::zip_iterator<Coord_Iterator_Tuple> Coord_3_Iterator;
    typedef thrust::pair<Coord_Iterator, Coord_Iterator> Coord_Iterator_Pair;
    typedef thrust::tuple<Coord_T, Coord_T, Coord_T> Coord3;
  public:

    Points_T(const int npts, thrust::host_vector<Coord_T> pts)
      : npts(npts) {
      // Create device vectors
      px = Coord_Vector(npts);
      py = Coord_Vector(npts);
      pz = Coord_Vector(npts);

      // Stride host vector for x, y, z
      typedef typename thrust::host_vector<Coord_T>::iterator Host_Itr;
      strided_range<Host_Itr> x(pts.begin(), pts.end(), 3);
      strided_range<Host_Itr> y(pts.begin()+1, pts.end(), 3);
      strided_range<Host_Itr> z(pts.begin()+2, pts.end(), 3);

      // Copy to device
      thrust::copy(x.begin(), x.end(), px.begin());
      thrust::copy(y.begin(), y.end(), py.begin());
      thrust::copy(z.begin(), z.end(), pz.begin());
    }

    // Take from zip iterator example
    Coord_3_Iterator begin() {
      return thrust::make_zip_iterator(make_tuple(px.begin(), py.begin(), pz.begin()));
    }

    Coord_3_Iterator end() {
      return thrust::make_zip_iterator(make_tuple(px.end(), py.end(), pz.end()));
    }

    Coord_Iterator_Pair x_minmax() {
      return thrust::minmax_element(px.begin(), px.end());
    }
    
    Coord_Iterator_Pair y_minmax() {
      return thrust::minmax_element(py.begin(), py.end());
    }
    
    Coord_Iterator_Pair z_minmax() {
      return thrust::minmax_element(pz.begin(), pz.end());
    }

    thrust::pair<array<Coord_T,3>, array<Coord_T,3> > minmax() {
      Coord_Iterator_Pair xpair = x_minmax();
      Coord_Iterator_Pair ypair = y_minmax();
      Coord_Iterator_Pair zpair = z_minmax();
      array<Coord_T,3> lo = {*thrust::get<0>(xpair), *thrust::get<0>(ypair), *thrust::get<0>(zpair)};
      array<Coord_T,3> hi = {*thrust::get<1>(xpair), *thrust::get<1>(ypair), *thrust::get<1>(zpair)};
      return thrust::pair<array<Coord_T,3>, array<Coord_T,3> >(lo, hi);
    }    

  private:
    int npts;
    Coord_Vector px;
    Coord_Vector py;
    Coord_Vector pz;
  };


  template<typename Coord_T>
  class Grid_T {
    typedef thrust::tuple<Coord_T, Coord_T, Coord_T> Coord3;

    typedef thrust::device_vector<Coord_T> Coord_Vector;
    typedef typename Coord_Vector::iterator Coord_Iterator;
    typedef thrust::tuple<Coord_Iterator, Coord_Iterator, Coord_Iterator> Coord_Iterator_Tuple;
    typedef thrust::zip_iterator<Coord_Iterator_Tuple> Coord_3_Iterator;

  public:
    int ng;
    int ng3;
    double r_cell;
    array<double,3> d_cell;
    int nfixpts;
    Points_T<Coord_T>* pts;
    thrust::device_vector<int> cells;
    thrust::device_vector<int> base;

    // Check if this is a legal cell.
    bool check(const Cell3 a) const {
      if (a[0] < 0 || a[0] >= ng ) return false;
      if (a[1] < 0 || a[1] >= ng ) return false;
      if (a[2] < 0 || a[2] >= ng ) return false;
      return true;
    }

    // clip is needed because roundoff errors may cause a number to be slightly outside the legal range. 
    void clip(Cell3 &a) {

      //    a[0] = min(ng-1, max(0, a[0]));

      if (a[0] < 0) a[0] = 0;
      if (a[0] >= ng) a[0] = ng-1;
      if (a[1] < 0) a[1] = 0;
      if (a[1] >= ng) a[1] = ng-1;
      if (a[2] < 0) a[2] = 0;
      if (a[2] >= ng) a[2] = ng-1;
    }
    
    // Compute_Cell_Containing_Point: Return the cell number containing point p.  Note the p is a
    // array<Coord_T, 3>, not an id of a point.  This is necessary since
    // Compute_Cell_Containing_Point is used for both fixed and query points.

    const Cell3 Compute_Cell_Containing_Point(const array<Coord_T,3> p) {
      const short int ix = static_cast<short int>(static_cast<double>(p[0])*r_cell+d_cell[0]);   // This must truncate not round.
      const short int iy = static_cast<short int>(static_cast<double>(p[1])*r_cell+d_cell[1]);
      const short int iz = static_cast<short int>(static_cast<double>(p[2])*r_cell+d_cell[2]);
      Cell3 c(ix, iy, iz);
      return c;
    }
    
    int point_to_id(const int& n) {
      Coord_3_Iterator p = pts->begin();
      int ix = static_cast<short int>(static_cast<double>(thrust::get<0>(p[n]))*r_cell+d_cell[0]);
      int iy = static_cast<short int>(static_cast<double>(thrust::get<1>(p[n]))*r_cell+d_cell[1]);
      int iz = static_cast<short int>(static_cast<double>(thrust::get<2>(p[n]))*r_cell+d_cell[2]);

      if (ix < 0 || ix >= ng || iy < 0 || iy >= ng || iz < 0 || iz >= ng) return -1;
      return (ix*ng + iy)*ng + iz;
    }

    int qpoint_to_id(const array<Coord_T, 3> q) {
      int ix = static_cast<short int>(static_cast<double>(q[0])*r_cell+d_cell[0]);
      int iy = static_cast<short int>(static_cast<double>(q[1])*r_cell+d_cell[1]);
      int iz = static_cast<short int>(static_cast<double>(q[2])*r_cell+d_cell[2]);

      if (ix < 0 || ix >= ng || iy < 0 || iy >= ng || iz < 0 || iz >= ng) return -1;
      return (ix*ng + iy)*ng + iz;
    }
    
    int cellid_to_int(const Cell3 a) const { 
      if (a[0]<0 || a[0] >=ng || a[1]<0 || a[1] >=ng || a[2]<0 || a[2] >=ng) return -1;
      return  (static_cast<int> (a[0])*ng + static_cast<int>(a[1]))*ng + a[2]; 
    }

    int num_points_id(const int id) {
      if (id<0) return 0;
      return base[id+1] - base[id];
    }

    void querythiscell(const Cell3 thiscell, const array<Coord_T, 3> q,
                       int &closestpt, double &dist2) {
      const int queryint(qpoint_to_id(q));
      const int npitc(num_points_id(queryint));
      if (npitc<=0) {
        closestpt = -1;
        dist2 = numeric_limits<double>::max();
        return;
      }
      typedef thrust::device_vector<int>::iterator IntItr;
      typedef thrust::permutation_iterator<Coord_3_Iterator, IntItr> PermItr;
      typedef thrust::transform_iterator<distance2_functor<Coord3>, PermItr> dist2_itr;
      PermItr ptsbegin(pts->begin(), cells.begin());
      dist2_itr begin(ptsbegin + base[queryint],
                      distance2_functor<Coord3>(q[0], q[1], q[2]));
      dist2_itr end(ptsbegin + base[queryint+1] - 1,
                    distance2_functor<Coord3>(q[0], q[1], q[2]));
      dist2_itr result = thrust::min_element(begin, end);
      closestpt = cells[result - begin + base[queryint]];
      dist2 = *result;
      return;
    }

    int Query_Fast_Case(const array<Coord_T, 3> q) {
      const int queryint(qpoint_to_id(q));
      const int npitc(num_points_id(queryint));

      #ifdef DEBUG
      cout << PRINTC(q[0]) << PRINTC(q[1]) << PRINTN(q[2]);
      cout << PRINTC(queryint) << PRINTN(npitc);
      cout << PRINTC(base[queryint]) << PRINTN(base[queryint+1]);
      #endif
      
      if (npitc<=0) return -1; // No points in this cell

      int closestpt = -1;

      // Thrust iterator black magic
      typedef thrust::device_vector<int>::iterator IntItr;
      typedef thrust::permutation_iterator<Coord_3_Iterator, IntItr> PermItr;
      typedef thrust::transform_iterator<distance2_functor<Coord3>, PermItr> dist2_itr;
      PermItr ptsbegin(pts->begin(), cells.begin());
      dist2_itr begin(ptsbegin + base[queryint],
                      distance2_functor<Coord3>(q[0], q[1], q[2]));
      dist2_itr end(ptsbegin + base[queryint+1] - 1,
                    distance2_functor<Coord3>(q[0], q[1], q[2]));
      
      Coord_3_Iterator i(pts->begin());
      i += cells[base[queryint]];
      Coord_T x = thrust::get<0>(*i);
      Coord_T y = thrust::get<1>(*i);
      Coord_T z = thrust::get<2>(*i);
      #ifdef DEBUG
      cout << PRINTC(cells[base[queryint]]) << PRINTN(cells[base[queryint+1]-1]);
      cout << PRINTC(*begin) << PRINTN(*end);
      cout << PRINTC(x) << PRINTC(y) << PRINTN(z);
      #endif
      
      dist2_itr result = thrust::min_element(begin, end);
      double dist2 = *result;

      closestpt = cells[result - begin + base[queryint]];
      //closestpt = cells[result - begin];
      
      const double distf = sqrt(dist2) * 1.00001;

      #ifdef DEBUG
      cout << PRINTC(closestpt) << PRINTC(end - begin) <<  PRINTC(result - begin) <<
        PRINTC(result - begin + base[queryint]) << PRINTC(*result) << PRINTN(distf);
      #endif

      array<Coord_T, 3> lopt, hipt;
      for (int i=0; i<3; ++i) {
        lopt[i] = static_cast<unsigned short int> (clamp_USI(static_cast<double>(q[i]) - distf));
        hipt[i] = static_cast<unsigned short int> (clamp_USI(static_cast<double>(q[i]) + distf + 1.0));
      }

      Cell3 locell(Compute_Cell_Containing_Point(lopt));
      Cell3 hicell(Compute_Cell_Containing_Point(hipt));

      clip(locell);
      clip(hicell);

      cout << PRINTC(lopt) << PRINTN(hipt);
      Cell3 qcell(Compute_Cell_Containing_Point(q));
      if (locell == qcell && hicell == qcell) return closestpt;

      
      for (Coord_T x=locell[0]; x<=hicell[0]; x++) {
        for (Coord_T y=locell[1]; y<=hicell[1]; y++) {
          // Do a whole z-row of cells at once.
          const int i01 = (static_cast<int>(x)*ng + static_cast<int>(y))*ng;
          const int i0 = i01 + locell[2];
          const int i1 = i01 + hicell[2];
          #ifdef DEBUG
          cout << PRINTC(x) << PRINTC(y) << PRINTC(i01) <<
            PRINTC(i0) << PRINTC(i1) << PRINTC(base[i0]) << PRINTN(base[i1+1]);
          #endif

          dist2_itr b(ptsbegin + base[i0],
                      distance2_functor<Coord3>(q[0], q[1], q[2]));
          cout << "A" << endl;
          dist2_itr e(ptsbegin + base[i1+1] - 1,
                      distance2_functor<Coord3>(q[0], q[1], q[2]));
          cout << "B" << endl;
          cout << PRINTC(*b) << PRINTN(*e);
          dist2_itr r = thrust::min_element(b, e);
          double d2 = *r;
          cout << PRINTC(base[i1+1]-1) << PRINTC(d2) << PRINTC(r - b) << PRINTC(e - b) << PRINTN(r - b + base[i0]);
          if (d2 < dist2 || (d2==dist2 && cells[r - b + base[i0]]<closestpt)) {
            cout << "C" << endl;
            dist2 = d2;
              closestpt = cells[r - b + base[i0]];
              cout << PRINTC(d2) << PRINTC(r - b + base[i0]) << PRINTN(cells[r - b + base[i0]]);
              cout << "D" << endl;
          }
        }
      }

      cout << PRINTC(closestpt) << PRINTN(dist2);
      cout << endl;
    
      return closestpt;
    }
  };
  

  template<typename Coord_T> Grid_T<Coord_T>*
  Preprocess(const int nfixpts, Points_T<Coord_T>* pts) {
    typedef thrust::tuple<Coord_T, Coord_T, Coord_T> Coord3;
    
    Grid_T<Coord_T> *g;
    g = new Grid_T<Coord_T>;
    g->nfixpts = nfixpts;
    int &ng = g->ng;
    ng = static_cast<int> (ng_factor * cbrt(static_cast<double>(nfixpts)));

    ng = min(2000, max(1, ng));
    g->ng3 = ng * ng * ng;
    g->pts = pts;

    for (int i=1; i<ncellsearchorder; ++i)
      if (nearpt3::cellsearchorder[i-1][3] > nearpt3::cellsearchorder[i][3]) 
        throw "cellsearchorder is not monotonic";

    thrust::pair<array<Coord_T,3>, array<Coord_T,3> > minmax = pts->minmax();
    array<Coord_T,3> lo = thrust::get<0>(minmax);
    array<Coord_T,3> hi = thrust::get<1>(minmax);

    #ifdef DEBUG
    cout << "Min/Max" << endl;
    cout << lo[0] << ", " << lo[1] << ", " << lo[2] << endl;
    cout << hi[0] << ", " << hi[1] << ", " << hi[2] << endl;
    #endif

    array<double,3> s;
    for (int i=0; i<3; ++i) {
      s[i] = 0.99 * ng / static_cast<double>(hi[i] - lo[i]);
    }
    g->r_cell = min(min(s[0], s[1]), s[2]);

    for(int i=0; i<3; i++) {
      g->d_cell[i] = ((ng-1)-(lo[i]+hi[i])*g->r_cell) * 0.5;
    }

    #ifdef DEBUG
    cout << "Grid info:";
    cout << "\nng: " << g->ng;
    cout << "\nng3: " << g->ng3;
    cout << "\ns: (" << s[0] << ", " << s[1] << ", " << s[2] << ")";
    cout << "\nr_cell: " << g->r_cell;
    cout << "\nd_cell: " << g->d_cell[0] << ", " << g->d_cell[1] << ", " << g->d_cell[2] << ")";
    cout << endl;
    #endif

    g->base = thrust::device_vector<int>(g->ng3+1, 1);
    g->cells = thrust::device_vector<int>(g->nfixpts);

    // Calculate cell id from point
    thrust::transform(pts->begin(), pts->end(), g->cells.begin(),
                      point_to_id_functor<Coord3>(g->ng, g->r_cell, g->d_cell[0],
                                                  g->d_cell[1], g->d_cell[2]));

    #ifdef DEBUG
    cout << "Cell IDs (cells): [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    // Ensure no cells are -1 (outside range)
    if (thrust::find(g->cells.begin(), g->cells.end(), -1) != g->cells.end()) {
      throw "Bad cell";
    }

    thrust::sort(g->cells.begin(), g->cells.end());

    #ifdef DEBUG
    cout << "Sorted Cell IDs (cells): [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    // Taken from thrust histogram example
    thrust::counting_iterator<int> count(0);
    thrust::lower_bound(g->cells.begin(), g->cells.end(),
                        count, count + g->ng3 + 1,
                        g->base.begin());

    #ifdef DEBUG
    cout << "Count: " << *count << endl;
    cout << "Lower bound (base): [";
    thrust::copy(g->base.begin(), g->base.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    if (g->base[g->ng3] != nfixpts) {
      cout << "ERROR: Internal inconsistency; wrong " << PRINTN(g->base[g->ng3]);
      throw "Internal inconsistency";
    }
    
    thrust::fill(g->cells.begin(), g->cells.end(), 0);

    // SERIAL
    for (int n=0; n<g->nfixpts; ++n) {
      const int ic(g->point_to_id(n));
      const int pitc = g->cells[g->base[ic+1]-1]++;
      g->cells[g->base[ic]+pitc] = n;
    }
    /*
    thrust::transform(pts->begin(), pts->end(), g->cells.begin(),
                      point_to_id_functor<Coord3>(g->ng, g->r_cell, g->d_cell[0],
                                                  g->d_cell[1], g->d_cell[2]));
    thrust::stable_sort_by_key();
    */
    #ifdef DEBUG
    cout << "Iterative (cells): [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    return g;
  }

  template<typename Coord_T> int
  Query(Grid_T<Coord_T>* g, const array<Coord_T, 3> q) {

    int closestpt(g->Query_Fast_Case(q));
    if (closestpt>=0) {
      return closestpt;
    }

    Cell3 querycell(g->Compute_Cell_Containing_Point(q));

    double dist(numeric_limits<double>::max());
    int closecell(-1);
    int goodsortnum;
    bool foundit(false);
    int nstop(ncellsearchorder);
    
    for (int isort=0; isort<nstop; ++isort) {
      int thisclosest;
      double thisdist;
      Cell3 s (cellsearchorder[isort][0], cellsearchorder[isort][1], 
	       cellsearchorder[isort][2]);

      for (int isign=0; isign<8; isign++) {      // Iterate over all combinations of signs;
        static const int sign3[8][3] = {{1,1,1},{1,1,-1},{1,-1,1},{1,-1,-1},
                                        {-1,1,1},{-1,1,-1},{-1,-1,1},{-1,-1,-1}};
        if (s[0]==0 && sign3[isign][0]== -1) continue;
        if (s[1]==0 && sign3[isign][1]== -1) continue;
        if (s[2]==0 && sign3[isign][2]== -1) continue;

        const Cell3 s2(s*sign3[isign]);

        for (int iperm=0; iperm<6; iperm++) {   // Iterate over all permutations of coordinates.
          switch (iperm) {
          case 1:
            if (s[1]==s[2]) continue;
            break;
          case 2: 
            if (s[0]==s[1]) continue;
            break;
          case 3:
          case 4:
            if (s[0]==s[1] && s[0]==s[2]) continue;
            break;
          case 5:
            if (s[0]==s[2]) continue;
            break;
          }
          static const int perm3[6][3] = {{0,1,2},{0,2,1},{1,0,2},{1,2,0},{2,0,1},{2,1,0}};
          const Cell3 s3(s2[perm3[iperm][0]], s2[perm3[iperm][1]], s2[perm3[iperm][2]]);
          const Cell3 c2(querycell+s3);
          if (!g->check(c2)) continue;  // outside the universe?
          goodsortnum = isort;
          g->querythiscell(c2, q, thisclosest, thisdist);
          if (thisclosest < 0) continue;

          // If two fixed points are the same distance from the query, then return the one with the
          // smallest index.  This removes ambiguities, but complicates the code in several places.
          
          if (thisdist<dist || (thisdist==dist && thisclosest<closestpt)) {
            dist = thisdist;
            closestpt = thisclosest;
            closecell =  g->cellid_to_int(c2);
            if (!foundit) {
              foundit = true;
              nstop = cellsearchorder[isort][3];
              if (nstop >= ncellsearchorder) {
                // It took so long to find any cell with a point that cellsearchorder doesn't have
                // enough cells to be sure of finding the closest point.  Fall back to naive
                // exhaustive searching.
                goto L_end_isort;
              }
            }
          }
        }
      }
    }

  L_end_isort: if (closestpt>=0) {
      return closestpt;
    }
    
    // No nearby points, so exhaustively search over all the fixed points.
    typedef thrust::tuple<Coord_T, Coord_T, Coord_T> Coord3;
    typedef thrust::device_vector<Coord_T> Coord_Vector;
    typedef typename Coord_Vector::iterator Coord_Iterator;
    typedef thrust::tuple<Coord_Iterator, Coord_Iterator, Coord_Iterator> Coord_Iterator_Tuple;
    typedef thrust::zip_iterator<Coord_Iterator_Tuple> Coord_3_Iterator;
    typedef thrust::device_vector<int>::iterator IntItr;
    typedef thrust::transform_iterator<distance2_functor<Coord3>, Coord_3_Iterator> dist2_itr;
    dist2_itr begin(g->pts->begin(),
                    distance2_functor<Coord3>(q[0], q[1], q[2]));
    dist2_itr end(g->pts->end(),
                    distance2_functor<Coord3>(q[0], q[1], q[2]));
    dist2_itr result = thrust::min_element(begin, end);
    closestpt = g->cells[result - begin];
    
    return closestpt;
  }
  
};

template<typename Coord_T> 
ostream &operator<<(ostream &o, const array<Coord_T,3> &c) {
  o << '(' << c[0] << ',' << c[1] << ',' << c[2] << ')';
  return o;
}
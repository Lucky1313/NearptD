#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/copy.h>

#include <algorithm>
#include <boost/multi_array.hpp> 
#include <iomanip>
#include <iostream>
#include <fstream>
#include <math.h>
#include <sys/resource.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <sys/times.h>
#include <sys/types.h>
#include <time.h>
#include <unistd.h>

using namespace std;
using boost::array;

#define DEBUG

// Print an expression's name then its value, possibly followed by a comma or endl.  
// Ex: cout << PRINTC(x) << PRINTN(y);

#define PRINT(arg)  #arg "=" << (arg)
#define PRINTC(arg)  #arg "=" << (arg) << ", "
#define PRINTN(arg)  #arg "=" << (arg) << endl

// From thrust example code
template <typename Iterator>
class strided_range
{
    public:

    typedef typename thrust::iterator_difference<Iterator>::type difference_type;

    struct stride_functor : public thrust::unary_function<difference_type,difference_type>
    {
        difference_type stride;

        stride_functor(difference_type stride)
            : stride(stride) {}

        __host__ __device__
        difference_type operator()(const difference_type& i) const
        { 
            return stride * i;
        }
    };

    typedef typename thrust::counting_iterator<difference_type>                   CountingIterator;
    typedef typename thrust::transform_iterator<stride_functor, CountingIterator> TransformIterator;
    typedef typename thrust::permutation_iterator<Iterator,TransformIterator>     PermutationIterator;

    // type of the strided_range iterator
    typedef PermutationIterator iterator;

    // construct strided_range for the range [first,last)
    strided_range(Iterator first, Iterator last, difference_type stride)
        : first(first), last(last), stride(stride) {}
   
    iterator begin(void) const
    {
        return PermutationIterator(first, TransformIterator(CountingIterator(0), stride_functor(stride)));
    }

    iterator end(void) const
    {
        return begin() + ((last - first) + (stride - 1)) / stride;
    }
    
    protected:
    Iterator first;
    Iterator last;
    difference_type stride;
};

template<typename T>
struct point_to_id_functor : public thrust::unary_function<T, int>
{
  const int ng;
  const double r_cell;
  const double d1;
  const double d2;
  const double d3;

  point_to_id_functor(int ng, double r_cell, double d1, double d2, double d3)
    : ng(ng), r_cell(r_cell), d1(d1), d2(d2), d3(d3) { }
  
  __host__ __device__
  int operator()(const T& a) const{
    int ix = static_cast<short int>(static_cast<double>(thrust::get<0>(a))*r_cell+d1);
    int iy = static_cast<short int>(static_cast<double>(thrust::get<1>(a))*r_cell+d2);
    int iz = static_cast<short int>(static_cast<double>(thrust::get<2>(a))*r_cell+d3);

    if (ix < 0 || ix >= ng || iy < 0 || iy >= ng || iz < 0 || iz >= ng) return -1;
    return (ix*ng + iy)*ng + iz;
  }
};

template <typename T>
void write(ostream &o, const thrust::tuple<T, T, T>& c) {
  o << "(" << thrust::get<0>(c) << "," << thrust::get<1>(c) << "," << thrust::get<2>(c) << ")";
}

template <typename T>
void write(ostream &o, const array<T,3>& c) {
    o << "(" << c[0] << "," << c[1] << "," << c[2] << ")";
}


namespace nearpt3 {
	double ng_factor = 1.6;

  // cellsearchorder:
  // First 3 elements of each row:  the order in which to search (one 48th-ant of the) cells adjacent to the current cell.
  // 4th element:   where, in cellsearchorder, to stop searching after the first point is found.
  const static int  cellsearchorder[][4] = {
#include "cellsearchorder"
  };
  // Number of cells in cellsearchorder (before expanding symmetries).
  const static int ncellsearchorder = 
    sizeof(nearpt3::cellsearchorder) / sizeof(nearpt3::cellsearchorder[0][0])/4;

  typedef short int Cell3_Index_T;
  class Cell3 {
  public:
    array<Cell3_Index_T,3> c;

    Cell3(const Cell3_Index_T x, const Cell3_Index_T y, const Cell3_Index_T z) {
      c[0] = x; c[1] = y; c[2] = z; }

    Cell3(const Cell3 &a) { c[0] = a[0]; c[1] = a[1]; c[2] = a[2]; }

    Cell3() { c[0] = -1; c[1] = -1; c[2] = -1; }

    Cell3_Index_T & operator[] (const int i)  {  return c[i];  }

    const Cell3_Index_T & operator[] (const int i) const {  return c[i];  }

    const Cell3 operator+(const Cell3 &d) const {
      Cell3 r;
      r[0] = c[0]+d[0];
      r[1] = c[1]+d[1];
      r[2] = c[2]+d[2];
      return r;
    }

    const Cell3 operator*(const int *d) const {
      Cell3 r;
      r[0] = c[0]*d[0];
      r[1] = c[1]*d[1];
      r[2] = c[2]*d[2];
      return r;
    }

    bool operator==(const Cell3 &d)  const {
      return c[0]==d[0] && c[1]==d[1] && c[2]==d[2];
    }

    //    const Cell3 operator*(const int *) const;
  };

  void write(ostream &o, const Cell3& c) {
    o << '(' << c[0] << ',' << c[1] << ',' << c[2] << ") ";
  }

  template<typename Coord_T>
  class Points_T {
    // Convenience Typedefs
    typedef thrust::device_vector<Coord_T> Coord_Vector;
    typedef typename Coord_Vector::iterator Coord_Iterator;
    typedef thrust::tuple<Coord_Iterator, Coord_Iterator, Coord_Iterator> Coord_Iterator_Tuple;
    typedef thrust::zip_iterator<Coord_Iterator_Tuple> Coord_3_Iterator;
    typedef thrust::pair<Coord_Iterator, Coord_Iterator> Coord_Iterator_Pair;
    typedef thrust::tuple<Coord_T, Coord_T, Coord_T> Coord3;
  public:

    Points_T(const int npts, thrust::host_vector<Coord_T> pts)
      : npts(npts) {
      // Create device vectors
      px = Coord_Vector(npts);
      py = Coord_Vector(npts);
      pz = Coord_Vector(npts);

      // Stride host vector for x, y, z
      typedef typename thrust::host_vector<Coord_T>::iterator Host_Itr;
      strided_range<Host_Itr> x(pts.begin(), pts.end(), 3);
      strided_range<Host_Itr> y(pts.begin()+1, pts.end(), 3);
      strided_range<Host_Itr> z(pts.begin()+2, pts.end(), 3);

      // Copy to device
      thrust::copy(x.begin(), x.end(), px.begin());
      thrust::copy(y.begin(), y.end(), py.begin());
      thrust::copy(z.begin(), z.end(), pz.begin());
    }

    // Take from zip iterator example
    Coord_3_Iterator begin() {
      return thrust::make_zip_iterator(make_tuple(px.begin(), py.begin(), pz.begin()));
    }

    Coord_3_Iterator end() {
      return thrust::make_zip_iterator(make_tuple(px.end(), py.end(), pz.end()));
    }

    Coord_Iterator_Pair x_minmax() {
      return thrust::minmax_element(px.begin(), px.end());
    }
    
    Coord_Iterator_Pair y_minmax() {
      return thrust::minmax_element(py.begin(), py.end());
    }
    
    Coord_Iterator_Pair z_minmax() {
      return thrust::minmax_element(pz.begin(), pz.end());
    }

    thrust::pair<array<Coord_T,3>, array<Coord_T,3> > minmax() {
      Coord_Iterator_Pair xpair = x_minmax();
      Coord_Iterator_Pair ypair = y_minmax();
      Coord_Iterator_Pair zpair = z_minmax();
      array<Coord_T,3> lo = {*thrust::get<0>(xpair), *thrust::get<0>(ypair), *thrust::get<0>(zpair)};
      array<Coord_T,3> hi = {*thrust::get<1>(xpair), *thrust::get<1>(ypair), *thrust::get<1>(zpair)};
      return thrust::pair<array<Coord_T,3>, array<Coord_T,3> >(lo, hi);
    }    

  private:
    int npts;
    Coord_Vector px;
    Coord_Vector py;
    Coord_Vector pz;
  };

  

  template<typename Coord_T>
  class Grid_T {
    typedef thrust::tuple<Coord_T, Coord_T, Coord_T> Coord3;

    typedef thrust::device_vector<Coord_T> Coord_Vector;
    typedef typename Coord_Vector::iterator Coord_Iterator;
    typedef thrust::tuple<Coord_Iterator, Coord_Iterator, Coord_Iterator> Coord_Iterator_Tuple;
    typedef thrust::zip_iterator<Coord_Iterator_Tuple> Coord_3_Iterator;
    typedef thrust::pair<Coord_Iterator, Coord_Iterator> Coord_Iterator_Pair;
  public:
    int ng;
    int ng3;
    double r_cell;
    array<double,3> d_cell;
    int nfixpts;
    Points_T<Coord_T>* pts;
    thrust::device_vector<int> cells;
    thrust::device_vector<int> base;

    int point_to_id(const int& n) {
      Coord_3_Iterator p = pts->begin();
      int ix = static_cast<short int>(static_cast<double>(thrust::get<0>(p[n]))*r_cell+d_cell[0]);
      int iy = static_cast<short int>(static_cast<double>(thrust::get<1>(p[n]))*r_cell+d_cell[1]);
      int iz = static_cast<short int>(static_cast<double>(thrust::get<2>(p[n]))*r_cell+d_cell[2]);

      if (ix < 0 || ix >= ng || iy < 0 || iy >= ng || iz < 0 || iz >= ng) return -1;
      return (ix*ng + iy)*ng + iz;
    }
    
  };
  

  template<typename Coord_T> Grid_T<Coord_T>*
  Preprocess(const int nfixpts, Points_T<Coord_T>* pts) {
    typedef thrust::tuple<Coord_T, Coord_T, Coord_T> Coord3;
    
    Grid_T<Coord_T> *g;
    g = new Grid_T<Coord_T>;
    g->nfixpts = nfixpts;
    int &ng = g->ng;
    ng = static_cast<int> (ng_factor * cbrt(static_cast<double>(nfixpts)));

    ng = min(2000, max(1, ng));
    g->ng3 = ng * ng * ng;
    g->pts = pts;

    for (int i=1; i<ncellsearchorder; ++i)
      if (nearpt3::cellsearchorder[i-1][3] > nearpt3::cellsearchorder[i][3]) 
	throw "cellsearchorder is not monotonic";

    thrust::pair<array<Coord_T,3>, array<Coord_T,3> > minmax = pts->minmax();
    array<Coord_T,3> lo = thrust::get<0>(minmax);
    array<Coord_T,3> hi = thrust::get<1>(minmax);

    #ifdef DEBUG
    cout << "Min/Max" << endl;
    cout << lo[0] << ", " << lo[1] << ", " << lo[2] << endl;
    cout << hi[0] << ", " << hi[1] << ", " << hi[2] << endl;
    #endif

    array<double,3> s;
    array<double,3> d;
    
    for (int i=0; i<3; ++i) {
      s[i] = 0.99 * ng / static_cast<double>(hi[i] - lo[i]);
    }
    g->r_cell = min(min(s[0], s[1]), s[2]);

    for(int i=0; i<3; i++) {
      g->d_cell[i] = ((ng-1)-(lo[i]+hi[i])*g->r_cell) * 0.5;
    }

    #ifdef DEBUG
    cout << "Grid info:";
    cout << "\nng: " << g->ng;
    cout << "\nng3: " << g->ng3;
    cout << "\ns: (" << s[0] << ", " << s[1] << ", " << s[2] << ")";
    cout << "\nr_cell: " << g->r_cell;
    cout << "\nd_cell: " << g->d_cell[0] << ", " << g->d_cell[1] << ", " << g->d_cell[2] << ")";
    cout << endl;
    #endif

    g->base = thrust::device_vector<int>(g->ng3+1, 1);
    g->cells = thrust::device_vector<int>(g->nfixpts);

    double aaa[3];
    aaa[0] = g->d_cell[0];
    aaa[1] = g->d_cell[1];
    aaa[2] = g->d_cell[2];
    // Calculate cell id from point
    thrust::transform(pts->begin(), pts->end(), g->cells.begin(),
                      point_to_id_functor<Coord3>(g->ng, g->r_cell, aaa[0], aaa[1], aaa[2]));

    #ifdef DEBUG
    cout << "Cell IDs (cells): [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    // Ensure no cells are -1 (outside range)
    if (thrust::find(g->cells.begin(), g->cells.end(), -1) != g->cells.end()) {
      throw "Bad cell";
    }

    thrust::sort(g->cells.begin(), g->cells.end());

    #ifdef DEBUG
    cout << "Sorted Cell IDs (cells): [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    
    // Taken from thrust histogram example
    thrust::counting_iterator<int> search(0);
    thrust::lower_bound(g->cells.begin(), g->cells.end(),
                        search, search + g->ng3 + 1,
                        g->base.begin());

    #ifdef DEBUG
    cout << "Count: " << *search << endl;
    cout << "Lower bound (base): [";
    thrust::copy(g->base.begin(), g->base.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    /*
    if (*g->base.end() != nfixpts) {
      cout << "Error, internal inconsistency; wrong " << PRINTN(g->base[g->ng3]);
      throw "Internal Inconsistency";
    }
    */

    /*
    thrust::adjacent_difference(g->cells.begin(), g->cells.end(), g->cells.begin());

    #ifdef DEBUG
    cout << "Adjacent difference (cells): [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    */
    
    thrust::fill(g->cells.begin(), g->cells.end(), 0);
    
    for (int n=0; n<g->nfixpts; ++n) {
      const int ic(g->point_to_id(n));
      const int pitc = g->cells[g->base[ic+1]-1]++;
      g->cells[g->base[ic]+pitc] = n;
    }

    /*
    #ifdef DEBUG
    cout << "Count (cells): [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    typedef thrust::device_vector<int>::iterator intiterator;
    //thrust::transform_iterator<ptif, intiterator> itr(pts->begin(), point_to_id_functor<Coord3>(g->ng, g->r_cell, aaa[0], aaa[1], aaa[2]));

    #ifdef DEBUG
    cout << "Transform: [";
    thrust::copy(thrust::make_transform_iterator(pts->begin(),
                                                      point_to_id_functor<Coord3>(g->ng, g->r_cell, g->d_cell[0], g->d_cell[1], g->d_cell[2])),
                 thrust::make_transform_iterator(pts->end(),
                                                 point_to_id_functor<Coord3>(g->ng, g->r_cell, g->d_cell[0], g->d_cell[1], g->d_cell[2])),
                 ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    thrust::transform(g->cells.begin(), g->cells.end(),
                      thrust::make_transform_iterator(pts->begin(),
                                                      point_to_id_functor<Coord3>(g->ng, g->r_cell, g->d_cell[0], g->d_cell[1], g->d_cell[2])),
                      g->cells.begin(),
                      cell_to_point_functor(thrust::raw_pointer_cast(g->base.data())));

    #ifdef DEBUG
    cout << "(cells): [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif
    */

    
    #ifdef DEBUG
    cout << "Iterative (cells): [";
    thrust::copy(g->cells.begin(), g->cells.end(), ostream_iterator<int>(cout, ", "));
    cout << "]" << endl;
    #endif

    return g;
  }

  template<typename Coord_T> int
  Query(Grid_T<Coord_T>* g, const array<Coord_T, 3> q) {
    
    
  }
  
};
